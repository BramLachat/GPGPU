
#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE	100000000

__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
	int i = threadIdx.x;

	if (i < n)
	{
		c[i] = a[i] + b[i];
	}
}

int main()
{
	int *a, *b, *c;

	hipMallocManaged(&a, SIZE * sizeof(int));
	hipMallocManaged(&b, SIZE * sizeof(int));
	hipMallocManaged(&c, SIZE * sizeof(int));

	for (int i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	VectorAdd <<<4, SIZE/4>>> (a, b, c, SIZE);

	hipDeviceSynchronize();

	//for (int i = 0; i < 10; ++i)
	printf("c[%d] = %d\n", SIZE - 1, c[SIZE - 1]);

	hipFree(a);
	hipFree(b);
	hipFree(c);

	return 0;
}