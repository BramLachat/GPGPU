#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <fstream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <hip/hip_cooperative_groups.h>
//#include <memory> //needed for smart pointers

#include "Mesh.h"
#include "parse_stl.h"
#include "RayTriangleIntersect.cuh"
#include "TriangleTriangleIntersect.cuh"

void rayTriangleIntersect(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void rayTriangleIntersect_v2(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void rayTriangleIntersect_v3(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void rayTriangle_BlockPerTriangle(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
//void rayTriangle_BlockPerTriangle_v2(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void handleCudaError(hipError_t cudaERR);
__global__ void startGPU();

std::ofstream outfile;
hipEvent_t start_event, stop_event;

int main(int argc, char* argv[]) {

	outfile.open("file.csv", std::ios::app);

	std::string stl_file_inside;
	std::string stl_file_outside;
	std::cout << "Enter filename of inside mesh:" << std::endl;
	std::cin >> stl_file_inside;
	std::cout << "Enter filename of outside mesh:" << std::endl;
	std::cin >> stl_file_outside;

	//Only reads STL-file in binary format!!!
	std::cout << "Reading files:" << std::endl;
	std::unique_ptr<Mesh> triangleMesh_Inside = stl::parse_stl(stl_file_inside);
	std::unique_ptr<Mesh> triangleMesh_Outside = stl::parse_stl(stl_file_outside);

	Vertex* V1 = triangleMesh_Outside->getVertexAtIndex(0);
	Vertex* V2 = triangleMesh_Outside->getVertexAtIndex(1);
	Vertex* V3 = triangleMesh_Outside->getVertexAtIndex(2);

	float xCenter = (V1->getCoordinates()[0] + V2->getCoordinates()[0] + V3->getCoordinates()[0])/3;
	float yCenter = (V1->getCoordinates()[1] + V2->getCoordinates()[1] + V3->getCoordinates()[1])/3;
	float zCenter = (V1->getCoordinates()[2] + V2->getCoordinates()[2] + V3->getCoordinates()[2])/3;

	float direction[3] = { xCenter, yCenter, zCenter };
	
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	std::cout << "direction = " << direction[0] << ", " << direction[1] << ", " << direction[2] << std::endl;

	rayTriangleIntersect(direction, triangleMesh_Inside, triangleMesh_Outside);	
	rayTriangleIntersect_v2(direction, triangleMesh_Inside, triangleMesh_Outside);
	rayTriangleIntersect_v3(direction, triangleMesh_Inside, triangleMesh_Outside);
	rayTriangle_BlockPerTriangle(direction, triangleMesh_Inside, triangleMesh_Outside);
	//rayTriangle_BlockPerTriangle_v2(direction, triangleMesh_Inside, triangleMesh_Outside);
	
	std::cout << "Press Enter to quit program!" << std::endl;
	std::cin.get();
	std::cin.get();

	outfile.close();
	return 0;
}

void rayTriangleIntersect(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement
	startGPU<<<1,1>>>();
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "\t\t\tStartup time GPU = " << transferDuration << " milliseconds" << std::endl;

	std::cout << "Transfering data from cpu to gpu!" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool inside = true;
	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	float3* insideOrigins = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideOrigins;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideOrigins, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideOrigins, insideOrigins, sizeInsideVertices, hipMemcpyHostToDevice));
	
	float* cudaDir;
	handleCudaError(hipMalloc((void**)&cudaDir, 3 * sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3 * sizeof(float), hipMemcpyHostToDevice));

	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = outerMesh->getNumberOfVertices() * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));
	
	int* intersectionsPerOrigin = new int[numberOfInsideVertices];
	int* cudaIntersectionsPerOrigin;
	handleCudaError(hipMalloc((void**)&cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int)));

	hipDeviceSynchronize();
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	int totalIntersections = 0;
	std::cout << "Kernel execution: rayTriangleIntersect" << std::endl;

	int numberOfBlocks = ((int)((numberOfOutsideTriangles + 127) / 128));

	hipEventRecord(start_event);
	Intersection::intersect_triangleGPU<<<numberOfBlocks,128>>>(cudaInsideOrigins, cudaDir, cudaOutsideTriangles, cudaOutsideVertices, numberOfInsideVertices, numberOfOutsideTriangles, cudaIntersectionsPerOrigin);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	handleCudaError(hipMemcpy(intersectionsPerOrigin, cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	int i = 0;
	while (i < numberOfInsideVertices && inside)
	{
		if (intersectionsPerOrigin[i] % 2 == 0) {
			inside = false;
		}
		i++;
	}

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	for (int i = 0; i < numberOfInsideVertices; i++)
	{
		totalIntersections += intersectionsPerOrigin[i];
	}

	hipFree(cudaInsideOrigins);
	hipFree(cudaDir);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaIntersectionsPerOrigin);

	hipHostFree(insideOrigins);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);

	delete intersectionsPerOrigin;

	std::string result;
	if (inside) { result = "INSIDE"; }
	else { result = "OUTSIDE"; }
	std::cout << result << std::endl;
	outfile << std::to_string(milliseconds) + ";" + result + ";" + std::to_string(totalIntersections) + ";";
}

void rayTriangleIntersect_v2(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	std::cout << "Transfering data from cpu to gpu!" << std::endl;
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool inside = true;
	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	float3* insideOrigins = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideOrigins;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideOrigins, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideOrigins, insideOrigins, sizeInsideVertices, hipMemcpyHostToDevice));

	float* cudaDir;
	handleCudaError(hipMalloc((void**)&cudaDir, 3 * sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3 * sizeof(float), hipMemcpyHostToDevice));

	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = outerMesh->getNumberOfVertices() * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	int* intersectionsPerOrigin = new int[numberOfInsideVertices];
	int* cudaIntersectionsPerOrigin;
	handleCudaError(hipMalloc((void**)&cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int)));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	int totalIntersections = 0;
	std::cout << "Kernel execution: rayTriangleIntersect_v2" << std::endl;

	int numberOfBlocks = ((int)((numberOfOutsideTriangles + 127) / 128));

	hipEventRecord(start_event);
	Intersection::intersect_triangleGPU_v2<< <numberOfBlocks, 128 >> > (cudaInsideOrigins, cudaDir, cudaOutsideTriangles, cudaOutsideVertices, numberOfInsideVertices, numberOfOutsideTriangles, cudaIntersectionsPerOrigin);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	handleCudaError(hipMemcpy(intersectionsPerOrigin, cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	int i = 0;
	while (i < numberOfInsideVertices && inside)
	{
		if (intersectionsPerOrigin[i] % 2 == 0) {
			inside = false;
		}
		i++;
	}

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	for (int i = 0; i < numberOfInsideVertices; i++)
	{
		totalIntersections += intersectionsPerOrigin[i];
	}

	hipFree(cudaInsideOrigins);
	hipFree(cudaDir);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaIntersectionsPerOrigin);

	hipHostFree(insideOrigins);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);

	delete intersectionsPerOrigin;

	std::string result;
	if (inside) { result = "INSIDE"; }
	else { result = "OUTSIDE"; }
	std::cout << result << std::endl;
	outfile << std::to_string(milliseconds) + ";" + result + ";" + std::to_string(totalIntersections) + ";";
}

void rayTriangleIntersect_v3(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	std::cout << "Transfering data from cpu to gpu!" << std::endl;
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool inside = true;
	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	float3* insideOrigins = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideOrigins;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideOrigins, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideOrigins, insideOrigins, sizeInsideVertices, hipMemcpyHostToDevice));

	float* cudaDir;
	handleCudaError(hipMalloc((void**)&cudaDir, 3 * sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3 * sizeof(float), hipMemcpyHostToDevice));

	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = outerMesh->getNumberOfVertices() * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	int* intersectionsPerOrigin = new int[numberOfInsideVertices];
	int* cudaIntersectionsPerOrigin;
	handleCudaError(hipMalloc((void**)&cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int)));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	int totalIntersections = 0;
	std::cout << "Kernel execution: rayTriangleIntersect_v3" << std::endl;

	int numberOfBlocks = ((int)((numberOfOutsideTriangles + 127) / 128));

	hipEventRecord(start_event);
	Intersection::intersect_triangleGPU_v3 << <numberOfBlocks, 128 >> > (cudaInsideOrigins, cudaDir, cudaOutsideTriangles, cudaOutsideVertices, numberOfInsideVertices, numberOfOutsideTriangles, cudaIntersectionsPerOrigin);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	handleCudaError(hipMemcpy(intersectionsPerOrigin, cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	int i = 0;
	while (i < numberOfInsideVertices && inside)
	{
		if (intersectionsPerOrigin[i] % 2 == 0) {
			inside = false;
		}
		i++;
	}

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	for (int i = 0; i < numberOfInsideVertices; i++)
	{
		totalIntersections += intersectionsPerOrigin[i];
	}

	hipFree(cudaInsideOrigins);
	hipFree(cudaDir);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaIntersectionsPerOrigin);

	hipHostFree(insideOrigins);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);

	delete intersectionsPerOrigin;

	std::string result;
	if (inside) { result = "INSIDE"; }
	else { result = "OUTSIDE"; }
	std::cout << result << std::endl;
	outfile << std::to_string(milliseconds) + ";" + result + ";" + std::to_string(totalIntersections) + ";";
}

void rayTriangle_BlockPerTriangle(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	std::cout << "Transfering data from cpu to gpu!" << std::endl;
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool inside = true;

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	float3* insideOrigins = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideOrigins;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideOrigins, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideOrigins, insideOrigins, sizeInsideVertices, hipMemcpyHostToDevice));

	float* cudaDir;
	handleCudaError(hipMalloc((void**)&cudaDir, 3 * sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3 * sizeof(float), hipMemcpyHostToDevice));

	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = outerMesh->getNumberOfVertices() * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	int* intersectionsPerOrigin = new int[numberOfInsideVertices];
	int* cudaIntersectionsPerOrigin;
	handleCudaError(hipMalloc((void**)&cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int)));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	int totalIntersections = 0;
	std::cout << "Kernel execution: rayTriangle_BlockPerTriangle" << std::endl;

	hipEventRecord(start_event);
	Intersection::intersect_triangleGPU_BlockPerTriangle << <numberOfOutsideTriangles, 128 >> > (cudaInsideOrigins, cudaDir, cudaOutsideTriangles, cudaOutsideVertices, numberOfInsideVertices, cudaIntersectionsPerOrigin);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	handleCudaError(hipMemcpy(intersectionsPerOrigin, cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	int i = 0;
	while (i < numberOfInsideVertices && inside)
	{
		if (intersectionsPerOrigin[i] % 2 == 0) {
			inside = false;
		}
		i++;
	}

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	for (int i = 0; i < numberOfInsideVertices; i++)
	{
		totalIntersections += intersectionsPerOrigin[i];
	}

	hipFree(cudaInsideOrigins);
	hipFree(cudaDir);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaIntersectionsPerOrigin);

	hipHostFree(insideOrigins);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);

	delete intersectionsPerOrigin;

	std::string result;
	if (inside) { result = "INSIDE"; }
	else { result = "OUTSIDE"; }
	std::cout << result << std::endl;
	outfile << std::to_string(milliseconds) + ";" + result + ";" + std::to_string(totalIntersections) + ";";
}

/*void rayTriangle_BlockPerTriangle_v2(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	std::cout << "Transfering data from cpu to gpu!" << std::endl;
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool inside = true;

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	float3* insideOrigins = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideOrigins;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideOrigins, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideOrigins, insideOrigins, sizeInsideVertices, hipMemcpyHostToDevice));

	float* cudaDir;
	handleCudaError(hipMalloc((void**)&cudaDir, 3 * sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3 * sizeof(float), hipMemcpyHostToDevice));

	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = outerMesh->getNumberOfVertices() * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	int* intersectionsPerOrigin = new int[numberOfInsideVertices];
	int* cudaIntersectionsPerOrigin;
	handleCudaError(hipMalloc((void**)&cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int)));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	int totalIntersections = 0;
	std::cout << "Kernel execution: rayTriangle_BlockPerTriangle_v2" << std::endl;

	hipEventRecord(start_event);
	Intersection::intersect_triangleGPU_BlockPerTriangle_v2 << <numberOfOutsideTriangles, 128 >> > (cudaInsideOrigins, cudaDir, cudaOutsideTriangles, cudaOutsideVertices, numberOfInsideVertices, cudaIntersectionsPerOrigin);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	handleCudaError(hipMemcpy(intersectionsPerOrigin, cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	int i = 0;
	while (i < numberOfInsideVertices && inside)
	{
		if (intersectionsPerOrigin[i] % 2 == 0) {
			inside = false;
		}
		i++;
	}

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	for (int i = 0; i < numberOfInsideVertices; i++)
	{
		totalIntersections += intersectionsPerOrigin[i];
	}

	hipFree(cudaInsideOrigins);
	hipFree(cudaDir);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaIntersectionsPerOrigin);

	hipHostFree(insideOrigins);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);

	delete intersectionsPerOrigin;

	std::string result;
	if (inside) { result = "INSIDE"; }
	else { result = "OUTSIDE"; }
	std::cout << result << std::endl;
	outfile << std::to_string(milliseconds) + ";" + result + ";" + std::to_string(totalIntersections) + "\n";
}*/

void handleCudaError(hipError_t cudaERR) {
	if (cudaERR != hipSuccess) {
		printf("CUDA ERROR : %s\n", hipGetErrorString(cudaERR));
	}
}

__global__ void startGPU() {
	printf("GPU ready!\n");
}