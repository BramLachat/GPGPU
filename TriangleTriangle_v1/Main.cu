#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
//#include <memory> //needed for smart pointers

#include "Mesh.h"
#include "parse_stl.h"
#include "RayTriangleIntersect.cuh"
#include "TriangleTriangleIntersect.cuh"

void rayTriangleIntersect(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void TriangleTriangleIntersect(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void handleCudaError(hipError_t cudaERR);
__global__ void startGPU();

int main(int argc, char* argv[]) {
	std::string stl_file_inside;
	std::string stl_file_outside;
	int RayTriangle;
	int CPU;
	std::cout << "Enter filename of inside mesh:" << std::endl;
	std::cin >> stl_file_inside;
	std::cout << "Enter filename of outside mesh:" << std::endl;
	std::cin >> stl_file_outside;
	std::cout << "0 = RayTriangleIntersection, 1 = TriangleTriangleIntersection" << std::endl;
	std::cin >> RayTriangle;

	if (argc == 2) {
		stl_file_inside = argv[1];
	}
	else if (argc > 2) {
		std::cout << "ERROR: Too many command line arguments" << std::endl;
	}

	auto t1 = std::chrono::high_resolution_clock::now(); //start time measurement

	//Only reads STL-file in binary format!!!
	std::cout << "Reading files:" << std::endl;
	std::unique_ptr<Mesh> triangleMesh_Inside = stl::parse_stl(stl_file_inside);
	std::unique_ptr<Mesh> triangleMesh_Outside = stl::parse_stl(stl_file_outside);

	auto t2 = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto time = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();
	std::cout << "Time = " << time << " milliseconds" << std::endl;

	std::cout << "STL HEADER = " << triangleMesh_Inside->getName() << std::endl;
	std::cout << "# triangles = " << triangleMesh_Inside->getNumberOfTriangles() << std::endl;
	std::cout << "# vertices = " << triangleMesh_Inside->getNumberOfVertices() << std::endl;

	//triangleMesh_Inside.schrijf();

	std::cout << "STL HEADER = " << triangleMesh_Outside->getName() << std::endl;
	std::cout << "# triangles = " << triangleMesh_Outside->getNumberOfTriangles() << std::endl;
	std::cout << "# vertices = " << triangleMesh_Outside->getNumberOfVertices() << std::endl;

	//triangleMesh_Outside.schrijf();

	Vertex* V1 = triangleMesh_Outside->getVertexAtIndex(0);
	Vertex* V2 = triangleMesh_Outside->getVertexAtIndex(1);
	Vertex* V3 = triangleMesh_Outside->getVertexAtIndex(2);

	float xCenter = (V1->getCoordinates()[0] + V2->getCoordinates()[0] + V3->getCoordinates()[0])/3;
	float yCenter = (V1->getCoordinates()[1] + V2->getCoordinates()[1] + V3->getCoordinates()[1])/3;
	float zCenter = (V1->getCoordinates()[2] + V2->getCoordinates()[2] + V3->getCoordinates()[2])/3;

	float direction[3] = { xCenter, yCenter, zCenter };
	//float direction[3] = { 1.0, 1.0, 1.0 };

	std::cout << "direction = " << direction[0] << ", " << direction[1] << ", " << direction[2] << std::endl;

	//auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	if (RayTriangle == 0)
	{
		std::cout << "CPU? (yes = 1, no = 0)" << std::endl;
		std::cin >> CPU;
		if (CPU == 1)
		{
			//2 opties om unique ptr mee te geven als argument aan een functie:
			//https://stackoverflow.com/questions/30905487/how-can-i-pass-stdunique-ptr-into-a-function
			triangleMesh_Outside->rayTriangleIntersect(direction, triangleMesh_Inside); // CPU version
		}
		rayTriangleIntersect(direction, triangleMesh_Inside, triangleMesh_Outside); // GPU version
	}
	else
	{
		std::cout << "CPU? (yes = 1, no = 0)" << std::endl;
		std::cin >> CPU;
		if (CPU == 1)
		{
			triangleMesh_Outside->triangleTriangleIntersect(triangleMesh_Inside); // CPU version
		}
		TriangleTriangleIntersect(triangleMesh_Inside, triangleMesh_Outside); // GPU version
	}	
	
	//auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	//auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	//std::cout << "Time = " << duration << "ms" << std::endl;

	std::cout << "Press Enter to quit program!" << std::endl;
	std::cin.get();
	std::cin.get();
	return 0;
}

void rayTriangleIntersect(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement
	startGPU<<<1,1>>>();
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "\t\t\tStartup time GPU = " << transferDuration << " milliseconds" << std::endl;

	std::cout << "\t\t\tCalculating intersections! (GPU)" << std::endl;
	std::cout << "--- Data Transfer ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)& cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	//nodig om in kernel te controleren dat aantal keer dat test wordt uitgevoerd <= is dan het aantal driehoeken
	//int numberOfCudaCalculations = numberOfInsideVertices * numberOfOutsideTriangles;

	//bool* threadResult;
	//handleCudaError(hipMalloc((void**)& threadResult, numberOfCudaCalculations*sizeof(bool)));

	/*thrust::host_vector<float3> insideOrigins = innerMesh->getVerticesVector();
	thrust::device_vector<float3> cudaInsideOrigins(insideOrigins.size());
	cudaInsideOrigins = insideOrigins;
	thrust::device_ptr<float3> d_cudaInsideOrigins = cudaInsideOrigins.data();
	float3* deviceInsideOrigins = thrust::raw_pointer_cast(d_cudaInsideOrigins);*/

	float3* insideOrigins = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideOrigins;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	//handleCudaError(hipMallocManaged((void**)& cudaInsideOrigins, sizeInsideVertices));
	//memcpy(cudaInsideOrigins, insideOrigins, sizeInsideVertices);  waarschijnlijk moet ik zoiets ook nog doen bij hipHostAlloc???
	handleCudaError(hipMalloc((void**)& cudaInsideOrigins, sizeInsideVertices));
	handleCudaError(hipMemcpy(cudaInsideOrigins, insideOrigins, sizeInsideVertices, hipMemcpyHostToDevice));
	
	float* cudaDir;
	//handleCudaError(hipMallocManaged((void**)& cudaDir, 3*sizeof(float)));
	//memcpy(cudaDir, dir, 3 * sizeof(float));
	handleCudaError(hipMalloc((void**)& cudaDir, 3 * sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3*sizeof(float), hipMemcpyHostToDevice));

	/*thrust::host_vector<int3> outsideTriangles = outerMesh->getTrianglesVector();;
	thrust::device_vector<int3> cudaOutsideTriangles(outsideTriangles.size());
	cudaOutsideTriangles = outsideTriangles;
	thrust::device_ptr<int3> d_cudaOutsideTriangles = cudaOutsideTriangles.data();
	int3* deviceOutsideTriangles = thrust::raw_pointer_cast(d_cudaOutsideTriangles);*/

	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	//handleCudaError(hipMallocManaged((void**)& cudaOutsideTriangles, sizeOutsideTriangles));
	//memcpy(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles);
	handleCudaError(hipMalloc((void**)& cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpy(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	/*thrust::host_vector<float3> outsideVertices = outerMesh->getVerticesVector();
	thrust::device_vector<float3> cudaOutsideVertices(outsideVertices.size());
	cudaOutsideVertices = outsideVertices;
	thrust::device_ptr<float3> d_cudaOutsideVertices = cudaOutsideVertices.data();
	float3* deviceOutsideVertices = thrust::raw_pointer_cast(d_cudaOutsideVertices);*/

	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = outerMesh->getNumberOfVertices() * sizeof(float3);
	//handleCudaError(hipMallocManaged((void**)& cudaOutsideVertices, sizeOutsideVertices));
	//memcpy(cudaOutsideVertices, outsideVertices, sizeOutsideVertices);
	handleCudaError(hipMalloc((void**)& cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpy(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));
	
	//thrust::device_vector<int> intersectionsPerOrigin(numberOfInsideVertices);
	//int* d_intersectionsPerOrigin = thrust::raw_pointer_cast(&intersectionsPerOrigin[0]);
	/*int* intersectionsPerOrigin = new int[numberOfInsideVertices];
	int* cudaIntersectionsPerOrigin;
	handleCudaError(hipMalloc((void**)& cudaIntersectionsPerOrigin, numberOfInsideVertices*sizeof(int)));*/

	//thrust::device_vector<float3> resultVertices(numberOfInsideVertices);
	//float3* d_resultVertices = thrust::raw_pointer_cast(&resultVertices[0]);
	/*float3* resultVertices = new float3[numberOfInsideVertices];
	float3* cudaResultVertices;
	handleCudaError(hipMalloc((void**)& cudaResultVertices, numberOfInsideVertices * sizeof(float3)));*/

	int totalIntersections = 0;

	std::cout << "--- End Data Transfer ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "\t\t\tTime Data Transfer = " << transferDuration << " milliseconds" << std::endl;

	std::cout << "--- Calculating ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	int numberOfBlocks = ((int)((numberOfInsideVertices + 511) / 512));
	Intersection::intersect_triangleGPU<<<numberOfBlocks,512>>>(cudaInsideOrigins, cudaDir, cudaOutsideTriangles, cudaOutsideVertices, numberOfInsideVertices, numberOfOutsideTriangles, cudaInside);
	hipError_t err = hipGetLastError();
	handleCudaError(err);

	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));

	//std::vector<int> h_intersectionsPerOrigin(intersectionsPerOrigin.size());
	//hrust::copy(intersectionsPerOrigin.begin(), intersectionsPerOrigin.end(), h_intersectionsPerOrigin.begin());
	//handleCudaError(hipMemcpy(intersectionsPerOrigin, cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int), hipMemcpyDeviceToHost));

	//std::vector<float3> h_resultVertices(resultVertices.size());
	//thrust::copy(resultVertices.begin(), resultVertices.end(), h_resultVertices.begin());
	//handleCudaError(hipMemcpy(resultVertices, cudaResultVertices, numberOfInsideVertices * sizeof(float3), hipMemcpyDeviceToHost));

	std::cout << "--- End Calculating ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto calculatingDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	std::cout << "\t\t\tTime Calculating = " << calculatingDuration << " microseconds" << std::endl;
	//std::cout << "\t\t\tTotal Time GPU = " << calculatingDuration + transferDuration << "ms" << std::endl;

	/*std::cout << "Writing to file!" << std::endl;

	std::unique_ptr<std::vector<Vertex>> verticesToWrite = std::make_unique<std::vector<Vertex>>();
	verticesToWrite->reserve(numberOfInsideVertices);
	float x, y, z;
	for (int i = 0; i < numberOfInsideVertices; i++)
	{
		totalIntersections += intersectionsPerOrigin[i];
		x = resultVertices[i].x;
		y = resultVertices[i].y;
		z = resultVertices[i].z;
		if (x + y + z != 0)
		{
			verticesToWrite->emplace_back(x, y, z);
			inside = false;
		}
	}

	innerMesh->writeVerticesToFile(verticesToWrite, "OutsideVerticesCUDA.stl");*/

	hipFree(cudaInsideOrigins);
	hipFree(cudaDir);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaInside);
	/*hipFree(cudaIntersectionsPerOrigin);
	hipFree(cudaResultVertices);*/
	hipHostFree(insideOrigins);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	//delete insideOrigins;
	//delete outsideTriangles;
	//delete outsideVertices;
	/*delete intersectionsPerOrigin;
	delete resultVertices;*/

	std::cout << "totaal intersecties: " << totalIntersections << std::endl;
	if (*inside) { std::cout << "INSIDE" << std::endl; }
	else { std::cout << "OUTSIDE" << std::endl; }
	
	delete inside;
}

void TriangleTriangleIntersect(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement
	startGPU<<<1,1>>>();
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "\t\t\tStartup time GPU = " << transferDuration << " milliseconds" << std::endl;

	std::cout << "\t\t\tCalculating intersections! (GPU)" << std::endl;
	std::cout << "--- Data Transfer ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfOutsideVertices = outerMesh->getNumberOfVertices();
	int numberOfInsideTriangles = innerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	/* Alloceren en kopi�ren hoekpunten binnenste mesh naar GPU*/
	float3* insideVertices = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideVertices;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideVertices, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideVertices, insideVertices, sizeInsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken binnenste mesh naar GPU*/
	int3* insideTriangles = innerMesh->getInt3ArrayTriangles();
	int3* cudaInsideTriangles;
	int sizeInsideTriangles = numberOfInsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaInsideTriangles, sizeInsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaInsideTriangles, insideTriangles, sizeInsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren hoekpunten buitenste mesh naar GPU*/
	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = numberOfOutsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));
	
	/* Alloceren en kopi�ren driehoeken buitenste mesh naar GPU*/
	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren van geheugen op GPU om bij te houden met hoeveel driehoeken van de buitenste mesh deze ene driehoek van de binnenste mesh snijdt*/
	//int* intersectionsPerInsideTriangle = new int[numberOfInsideTriangles];
	//int* cudaIntersectionsPerInsideTriangle;
	//handleCudaError(hipMalloc((void**)&cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int)));

	/* Als deze waarde > 0 ==> De binnenste mesh ligt niet volledig in de buitenste mesh*/
	int totalIntersections = 0;

	std::cout << "--- End Data Transfer ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "\t\t\tTime Data Transfer = " << transferDuration << " milliseconds" << std::endl;

	std::cout << "--- Calculating ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	/* Uitvoeren CUDA kernel*/
	int numberOfBlocks = ((int)((numberOfInsideTriangles + 511) / 512));
	triangle_triangle_GPU<<<numberOfBlocks,512>>>(cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, numberOfOutsideTriangles);
	hipError_t err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));

	std::cout << "--- End Calculating ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto calculatingDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	std::cout << "\t\t\tTime Calculating = " << calculatingDuration << " microseconds" << std::endl;

	/*for (int i = 0; i < numberOfInsideTriangles; i++)
	{
		totalIntersections += intersectionsPerInsideTriangle[i];
	}
	if (totalIntersections > 0) { inside = false; }*/

	hipFree(cudaInsideTriangles);
	hipFree(cudaInsideVertices);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaInside);
	//hipFree(cudaIntersectionsPerInsideTriangle);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	hipHostFree(insideTriangles);
	hipHostFree(insideVertices);
	//delete intersectionsPerInsideTriangle;

	std::cout << "totaal intersecties: " << totalIntersections << std::endl;
	if (*inside) { std::cout << "SNIJDEN NIET" << std::endl; }
	else { std::cout << "SNIJDEN WEL" << std::endl; }

	delete inside;
}

void handleCudaError(hipError_t cudaERR) {
	if (cudaERR != hipSuccess) {
		printf("CUDA ERROR : %s\n", hipGetErrorString(cudaERR));
	}
}

__global__ void startGPU() {
	printf("GPU ready!\n");
}