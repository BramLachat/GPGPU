
#include <hip/hip_runtime.h>
/*#include <iostream>

static void HandleError(cudaError_t err,
	const char* file,
	int line) {
	if (err != cudaSuccess) {
		printf("%s in %s at line %d\n", cudaGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))*/

/*int main(void)
{
	cudaDeviceProp prop;

	int count;
	HANDLE_ERROR(cudaGetDeviceCount(&count));

	for (int i = 0; i < count; i++)
	{
		HANDLE_ERROR(cudaGetDeviceProperties(&prop,i));
		printf("Name: %s\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
	}
}*/