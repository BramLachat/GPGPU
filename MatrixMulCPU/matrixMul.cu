
#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE	1024

void VectorAdd(int *a, int *b, int *c, int n)
{
	int i;

	for (i = 0; i < n; ++i)
		c[i] = a[i] + b[i];
}

int main()
{
	int *a, *b, *c;

	a = (int *)malloc(SIZE * sizeof(int));
	b = (int *)malloc(SIZE * sizeof(int));
	c = (int *)malloc(SIZE * sizeof(int));

	for (int i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	VectorAdd(a, b, c, SIZE);

	//for (int i = 0; i < 10; ++i)
	printf("c[%d] = %d\n", SIZE-1, c[SIZE-1]);

	free(a);
	free(b);
	free(c);

	return 0;
}