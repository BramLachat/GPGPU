int numBlocks;        // Occupancy in terms of active blocks
	int blockSize = 128;//meegeven volgens welke configuratie ge uw kernel zou launchen

	// These variables are used to convert occupancy to warps
	int device;
	hipDeviceProp_t prop;
	int activeWarps;
	int maxWarps;

	hipGetDevice(&device);
	hipGetDeviceProperties(&prop, device);

	hipOccupancyMaxActiveBlocksPerMultiprocessor(
		&numBlocks,
		Intersection::intersect_triangleGPU,
		blockSize,
		0);

	activeWarps = numBlocks * blockSize / prop.warpSize;
	maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;

	std::cout << "Occupancy: " << (double)activeWarps / maxWarps * 100 << "%" << std::endl;