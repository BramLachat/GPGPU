#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
//#include <memory> //needed for smart pointers

#include "Mesh.h"
#include "parse_stl.h"
#include "RayTriangleIntersect.cuh"

void rayTriangleIntersect(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void handleCudaError(hipError_t cudaERR);

int main(int argc, char* argv[]) {
	std::string stl_file_inside;
	std::string stl_file_outside;
	int RayTriangle;
	std::cout << "Enter filename of inside mesh:" << std::endl;
	std::cin >> stl_file_inside;
	std::cout << "Enter filename of outside mesh:" << std::endl;
	std::cin >> stl_file_outside;
	std::cout << "0 = RayTriangleIntersection, 1 = TriangleTriangleIntersection" << std::endl;
	std::cin >> RayTriangle;

	if (argc == 2) {
		stl_file_inside = argv[1];
	}
	else if (argc > 2) {
		std::cout << "ERROR: Too many command line arguments" << std::endl;
	}

	auto t1 = std::chrono::high_resolution_clock::now(); //start time measurement

	//Only reads STL-file in binary format!!!
	std::cout << "lezen" << std::endl;
	std::unique_ptr<Mesh> triangleMesh_Inside = stl::parse_stl(stl_file_inside);
	std::unique_ptr<Mesh> triangleMesh_Outside = stl::parse_stl(stl_file_outside);

	auto t2 = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto time = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();
	std::cout << "Time = " << time << "ms" << std::endl;

	std::cout << "STL HEADER = " << triangleMesh_Inside->getName() << std::endl;
	std::cout << "# triangles = " << triangleMesh_Inside->getNumberOfTriangles() << std::endl;
	std::cout << "# vertices = " << triangleMesh_Inside->getNumberOfVertices() << std::endl;

	//triangleMesh_Inside.schrijf();

	std::cout << "STL HEADER = " << triangleMesh_Outside->getName() << std::endl;
	std::cout << "# triangles = " << triangleMesh_Outside->getNumberOfTriangles() << std::endl;
	std::cout << "# vertices = " << triangleMesh_Outside->getNumberOfVertices() << std::endl;

	//triangleMesh_Outside.schrijf();

	Vertex* V1 = triangleMesh_Outside->getVertexAtIndex(0);
	Vertex* V2 = triangleMesh_Outside->getVertexAtIndex(1);
	Vertex* V3 = triangleMesh_Outside->getVertexAtIndex(2);

	float xCenter = (V1->getCoordinates()[0] + V2->getCoordinates()[0] + V3->getCoordinates()[0])/3;
	float yCenter = (V1->getCoordinates()[1] + V2->getCoordinates()[1] + V3->getCoordinates()[1])/3;
	float zCenter = (V1->getCoordinates()[2] + V2->getCoordinates()[2] + V3->getCoordinates()[2])/3;

	float direction[3] = { xCenter, yCenter, zCenter };
	//float direction[3] = { 1.0, 1.0, 1.0 };

	std::cout << "direction = " << direction[0] << ", " << direction[1] << ", " << direction[2] << std::endl;

	//auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	if (RayTriangle == 0)
	{
		auto start = std::chrono::high_resolution_clock::now(); //start time measurement

		//2 opties om unique ptr mee te geven als argument aan een functie:
		//https://stackoverflow.com/questions/30905487/how-can-i-pass-stdunique-ptr-into-a-function
		triangleMesh_Outside->rayTriangleIntersect(direction, triangleMesh_Inside);

		auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
		auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
		std::cout << "Time = " << duration << "ms" << std::endl;

		start = std::chrono::high_resolution_clock::now(); //start time measurement

		rayTriangleIntersect(direction, triangleMesh_Inside, triangleMesh_Outside);

		end = std::chrono::high_resolution_clock::now(); //stop time measurement
		duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
		std::cout << "Time = " << duration << "ms" << std::endl;
	}
	else
	{
		triangleMesh_Outside->triangleTriangleIntersect(triangleMesh_Inside);
	}	
	
	//auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	//auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	//std::cout << "Time = " << duration << "ms" << std::endl;

	std::cout << "Press Enter to quit program!" << std::endl;
	std::cin.get();
	std::cin.get();
	return 0;
}

void rayTriangleIntersect(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	bool inside = true;
	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	//nodig om in kernel te controleren dat aantal keer dat test wordt uitgevoerd <= is dan het aantal driehoeken
	int numberOfCudaCalculations = numberOfInsideVertices;
	//handleCudaError(hipMalloc((void**)& numberOfCudaCalculations, sizeof(int)));
	//handleCudaError(hipMemcpy(numberOfCudaCalculations, &numberOfInsideVertices, sizeof(int), hipMemcpyHostToDevice));

	//float* orig;
	//float* cudaOrig;
	float* insideOrigins = innerMesh->getFloatArrayVertices();
	float* cudaInsideOrigins;
	int sizeInsideVertices = 3 * numberOfInsideVertices * sizeof(float);
	handleCudaError(hipMalloc((void**)& cudaInsideOrigins, sizeInsideVertices));
	handleCudaError(hipMemcpy(cudaInsideOrigins, insideOrigins, sizeInsideVertices, hipMemcpyHostToDevice));
	
	float* cudaDir;
	handleCudaError(hipMalloc((void**)& cudaDir, 3*sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3*sizeof(float), hipMemcpyHostToDevice));

	bool* result = new bool[numberOfInsideVertices];
	bool* cudaResult;
	handleCudaError(hipMalloc((void**)& cudaResult, numberOfInsideVertices * sizeof(bool)));

	int* outsideTriangles = outerMesh->getIntArrayTriangles();
	int* cudaOutsideTriangles;
	int sizeOutsideTriangles = 3 * numberOfOutsideTriangles * sizeof(int);
	handleCudaError(hipMalloc((void**)& cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpy(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	float* outsideVertices = outerMesh->getFloatArrayVertices();
	float* cudaOutsideVertices;
	int sizeOutsideVertices = 3 * outerMesh->getNumberOfVertices() * sizeof(float);
	handleCudaError(hipMalloc((void**)& cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpy(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	thrust::device_vector<int> intersectionsPerThread(numberOfInsideVertices);
	int* d_intersectionsPerThread = thrust::raw_pointer_cast(&intersectionsPerThread[0]);

	thrust::device_vector<float3> resultVertices(numberOfInsideVertices);
	float3* d_resultVertices = thrust::raw_pointer_cast(&resultVertices[0]);
	//float3* d_outsideVertices = thrust::raw_pointer_cast(&outsideVertices[0]); LUKT NIET!!!

	int totalIntersections = 0;
	/*
	for (int j = 0; j < innerMesh->getNumberOfVertices(); j++)
	{
		orig = (innerMesh->getVertexAtIndex(j))->getCoordinates();
		handleCudaError(hipMalloc((void**)& cudaOrig, 3 * sizeof(float)));
		handleCudaError(hipMemcpy(cudaOrig, orig, 3 * sizeof(float), hipMemcpyHostToDevice));

		int numberOfIntersections = 0;

		int numberOfBlocks = ((int)((numberOfTriangles+255)/256));

		Intersection::intersect_triangle4<<<numberOfBlocks,256>>>(cudaOrig, cudaDir, cudaTriangles, cudaVertices, cudaResult, numberOfCalculations);
		hipError_t err = hipGetLastError();
		handleCudaError(err);

		handleCudaError(hipMemcpy(result, cudaResult, numberOfTriangles * sizeof(int), hipMemcpyDeviceToHost));

		//std::cout << "result = ";
		for (int i = 0; i < numberOfTriangles; i++)
		{
			if (result[i] == 1) { numberOfIntersections++; }
			//std::cout << result[i] << ", ";
		}
		totalIntersections += numberOfIntersections;
		//std::cout << "numberOfIntersections: " << numberOfIntersections;
		//std::cout << std::endl;
		if (numberOfIntersections % 2 == 0)
		{
			inside = false;
		}
		//hipFree(cudaOrig);
	}*/
	int numberOfBlocks = ((int)((numberOfInsideVertices + 255) / 256));
	Intersection::intersect_triangleGPU<<<numberOfBlocks,256>>>(cudaInsideOrigins, cudaDir, cudaOutsideTriangles, cudaOutsideVertices, cudaResult, numberOfCudaCalculations, numberOfOutsideTriangles, d_intersectionsPerThread, d_resultVertices);
	hipError_t err = hipGetLastError();
	handleCudaError(err);

	handleCudaError(hipMemcpy(result, cudaResult, numberOfInsideVertices * sizeof(bool), hipMemcpyDeviceToHost));

	std::vector<int> h_intersectionsPerThread(intersectionsPerThread.size());
	thrust::copy(intersectionsPerThread.begin(), intersectionsPerThread.end(), h_intersectionsPerThread.begin());

	std::vector<float3> h_resultVertices(resultVertices.size());
	thrust::copy(resultVertices.begin(), resultVertices.end(), h_resultVertices.begin());
	std::unique_ptr<std::vector<Vertex>> verticesToWrite = std::make_unique<std::vector<Vertex>>();
	verticesToWrite->reserve(h_resultVertices.size());
	float x, y, z;
	for (int i = 0; i < h_resultVertices.size(); i++) 
	{
		x = h_resultVertices[i].x;
		y = h_resultVertices[i].y;
		z = h_resultVertices[i].z;
		if (x + y + z != 0) { verticesToWrite->emplace_back(x, y, z); }
	}
	innerMesh->writeVerticesToFile(verticesToWrite, "OutsideVerticesCUDA.stl");

	for (int i = 0; i < numberOfInsideVertices; i++)
	{
		if (!result[i])
		{
			inside = false;
		}
		totalIntersections += h_intersectionsPerThread[i];
	}

	hipFree(cudaInsideOrigins);
	hipFree(cudaDir);
	hipFree(cudaResult);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	//hipFree(numberOfCudaCalculations);
	delete result;
	delete insideOrigins;
	delete outsideTriangles;
	delete outsideVertices;

	std::cout << "totaal intersecties: " << totalIntersections << std::endl;
	if (inside) { std::cout << "INSIDE" << std::endl; }
	else { std::cout << "OUTSIDE" << std::endl; }
}

void handleCudaError(hipError_t cudaERR) {
	if (cudaERR != hipSuccess) {
		printf("CUDA ERROR : %s\n", hipGetErrorString(cudaERR));
	}
}