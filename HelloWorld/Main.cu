#include <cassert>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
//#include <memory> //needed for smart pointers

#include "Mesh.h"
#include "parse_stl.h"
#include "RayTriangleIntersect.cuh"

void rayTriangleIntersect(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void handleCudaError(hipError_t cudaERR);

int main(int argc, char* argv[]) {
	std::string stl_file_inside;
	std::string stl_file_outside;
	int RayTriangle;
	std::cout << "Enter filename of inside mesh:" << std::endl;
	std::cin >> stl_file_inside;
	std::cout << "Enter filename of outside mesh:" << std::endl;
	std::cin >> stl_file_outside;
	std::cout << "0 = RayTriangleIntersection, 1 = TriangleTriangleIntersection" << std::endl;
	std::cin >> RayTriangle;

	if (argc == 2) {
		stl_file_inside = argv[1];
	}
	else if (argc > 2) {
		std::cout << "ERROR: Too many command line arguments" << std::endl;
	}

	auto t1 = std::chrono::high_resolution_clock::now(); //start time measurement

	//Only reads STL-file in binary format!!!
	std::cout << "lezen" << std::endl;
	std::unique_ptr<Mesh> triangleMesh_Inside = stl::parse_stl(stl_file_inside);
	std::unique_ptr<Mesh> triangleMesh_Outside = stl::parse_stl(stl_file_outside);

	auto t2 = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto time = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();
	std::cout << "Time = " << time << "ms" << std::endl;

	std::cout << "STL HEADER = " << triangleMesh_Inside->getName() << std::endl;
	std::cout << "# triangles = " << triangleMesh_Inside->getNumberOfTriangles() << std::endl;
	std::cout << "# vertices = " << triangleMesh_Inside->getNumberOfVertices() << std::endl;

	//triangleMesh_Inside.schrijf();

	std::cout << "STL HEADER = " << triangleMesh_Outside->getName() << std::endl;
	std::cout << "# triangles = " << triangleMesh_Outside->getNumberOfTriangles() << std::endl;
	std::cout << "# vertices = " << triangleMesh_Outside->getNumberOfVertices() << std::endl;

	//triangleMesh_Outside.schrijf();

	Vertex* V1 = triangleMesh_Outside->getVertexAtIndex(0);
	Vertex* V2 = triangleMesh_Outside->getVertexAtIndex(1);
	Vertex* V3 = triangleMesh_Outside->getVertexAtIndex(2);

	float xCenter = (V1->getCoordinates()[0] + V2->getCoordinates()[0] + V3->getCoordinates()[0])/3;
	float yCenter = (V1->getCoordinates()[1] + V2->getCoordinates()[1] + V3->getCoordinates()[1])/3;
	float zCenter = (V1->getCoordinates()[2] + V2->getCoordinates()[2] + V3->getCoordinates()[2])/3;

	float direction[3] = { xCenter, yCenter, zCenter };
	//float direction[3] = { 1.0, 1.0, 1.0 };

	std::cout << "direction = " << direction[0] << ", " << direction[1] << ", " << direction[2] << std::endl;

	//auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	if (RayTriangle == 0)
	{
		auto start = std::chrono::high_resolution_clock::now(); //start time measurement

		//2 opties om unique ptr mee te geven als argument aan een functie:
		//https://stackoverflow.com/questions/30905487/how-can-i-pass-stdunique-ptr-into-a-function
		triangleMesh_Outside->rayTriangleIntersect(direction, triangleMesh_Inside);

		auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
		auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
		std::cout << "Time = " << duration << "ms" << std::endl;

		start = std::chrono::high_resolution_clock::now(); //start time measurement

		rayTriangleIntersect(direction, triangleMesh_Inside, triangleMesh_Outside);

		end = std::chrono::high_resolution_clock::now(); //stop time measurement
		duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
		std::cout << "Time = " << duration << "ms" << std::endl;
	}
	else
	{
		triangleMesh_Outside->triangleTriangleIntersect(triangleMesh_Inside);
	}	
	
	//auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	//auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	//std::cout << "Time = " << duration << "ms" << std::endl;

	std::cout << "Press Enter to quit program!" << std::endl;
	std::cin.get();
	std::cin.get();
	return 0;
}

void rayTriangleIntersect(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	bool inside = true;
	int numberOfTriangles = outerMesh->getNumberOfTriangles();

	//nodig om in kernel te controleren dat aantal keer dat test wordt uitgevoerd <= is dan het aantal driehoeken
	int* numberOfCalculations;
	handleCudaError(hipMalloc((void**)& numberOfCalculations, sizeof(int)));
	handleCudaError(hipMemcpy(numberOfCalculations, &numberOfTriangles, sizeof(int), hipMemcpyHostToDevice));

	float* orig;
	float* cudaOrig;
	
	float* cudaDir;
	handleCudaError(hipMalloc((void**)& cudaDir, 3*sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3*sizeof(float), hipMemcpyHostToDevice));

	int* result = new int[numberOfTriangles];
	int* cudaResult;
	handleCudaError(hipMalloc((void**)& cudaResult, numberOfTriangles * sizeof(int))); //dit hoeft geen lijst van int's te zijn boolean is voldoende

	int* triangles = outerMesh->getIntArrayTriangles();
	int* cudaTriangles;
	int sizeTriangles = 3 * numberOfTriangles * sizeof(int);
	handleCudaError(hipMalloc((void**)& cudaTriangles, sizeTriangles));
	handleCudaError(hipMemcpy(cudaTriangles, triangles, sizeTriangles, hipMemcpyHostToDevice));

	float* vertices = outerMesh->getFloatArrayVertices();
	float* cudaVertices;
	int sizeVertices = 3 * outerMesh->getNumberOfVertices() * sizeof(float);
	handleCudaError(hipMalloc((void**)& cudaVertices, sizeVertices));
	handleCudaError(hipMemcpy(cudaVertices, vertices, sizeVertices, hipMemcpyHostToDevice));

	int totalIntersections = 0;

	for (int j = 0; j < innerMesh->getNumberOfVertices(); j++)
	{
		orig = (innerMesh->getVertexAtIndex(j))->getCoordinates();
		handleCudaError(hipMalloc((void**)& cudaOrig, 3 * sizeof(float)));
		handleCudaError(hipMemcpy(cudaOrig, orig, 3 * sizeof(float), hipMemcpyHostToDevice));

		int numberOfIntersections = 0;

		int numberOfBlocks = ((int)((numberOfTriangles+255)/256));

		Intersection::intersect_triangle4<<<numberOfBlocks,256>>>(cudaOrig, cudaDir, cudaTriangles, cudaVertices, cudaResult, numberOfCalculations);
		hipError_t err = hipGetLastError();
		handleCudaError(err);

		handleCudaError(hipMemcpy(result, cudaResult, numberOfTriangles * sizeof(int), hipMemcpyDeviceToHost));

		//std::cout << "result = ";
		for (int i = 0; i < numberOfTriangles; i++)
		{
			if (result[i] == 1) { numberOfIntersections++; }
			//std::cout << result[i] << ", ";
		}
		totalIntersections += numberOfIntersections;
		//std::cout << "numberOfIntersections: " << numberOfIntersections;
		//std::cout << std::endl;
		if (numberOfIntersections % 2 == 0)
		{
			inside = false;
		}
		hipFree(cudaOrig);
	}
	hipFree(cudaDir);
	hipFree(cudaResult);
	hipFree(cudaTriangles);
	hipFree(cudaVertices);
	hipFree(numberOfCalculations);
	delete result;
	std::cout << "totaal intersecties: " << totalIntersections << std::endl;
	if (inside) { std::cout << "INSIDE" << std::endl; }
	else { std::cout << "OUTSIDE" << std::endl; }
}

void handleCudaError(hipError_t cudaERR) {
	if (cudaERR != hipSuccess) {
		printf("CUDA ERROR : %s\n", hipGetErrorString(cudaERR));
	}
}