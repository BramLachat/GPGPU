#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <fstream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
//#include <memory> //needed for smart pointers

#include "Mesh.h"
#include "parse_stl.h"
#include "RayTriangleIntersect.cuh"
#include "TriangleTriangleIntersect.cuh"

void rayTriangle_BlockPerOrigin(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void rayTriangle_ThreadPerOrigin(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void rayTriangle_ThreadPerTriangle(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);

void TriangleTriangle_ThreadPerInnerTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void TriangleTriangle_BlockPerInnerTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void TriangleTriangle_ThreadPerOuterTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);

void handleCudaError(hipError_t cudaERR);
__global__ void startGPU();
void writeResultsToFile(std::vector<std::string>& result);

/* Console output wegschrijven naar file*/
std::vector<std::string> output;

hipEvent_t start_event, stop_event;

int main(int argc, char* argv[]) {
	//output.push_back(";RT_v1(BPO)(ms);;;RT_v2(TPO)(ms);;;RT_v3(TPT)(ms);;;TT_v1(TPIT)(ms);;;TT_v2(BPIT)(ms);;;TT_v3(TPOT)(ms);;\n");

	std::string stl_file_inside;
	std::string stl_file_outside;
	std::cout << "Enter filename of inside mesh:" << std::endl;
	std::cin >> stl_file_inside;
	std::cout << "Enter filename of outside mesh:" << std::endl;
	std::cin >> stl_file_outside;


	std::string delimiter = "\\";	

	//Only reads STL-file in binary format!!!
	std::cout << "Reading files:" << std::endl;
	std::unique_ptr<Mesh> triangleMesh_Inside = stl::parse_stl(stl_file_inside);
	std::unique_ptr<Mesh> triangleMesh_Outside = stl::parse_stl(stl_file_outside);

	size_t pos = 0;
	std::string token;
	while ((pos = stl_file_inside.find(delimiter)) != std::string::npos) {
		token = stl_file_inside.substr(0, pos);
		stl_file_inside.erase(0, pos + delimiter.length());
	}
	stl_file_inside = stl_file_inside.substr(0, stl_file_inside.find(".stl"));

	pos = 0;
	while ((pos = stl_file_outside.find(delimiter)) != std::string::npos) {
		token = stl_file_outside.substr(0, pos);
		stl_file_outside.erase(0, pos + delimiter.length());
	}
	stl_file_outside = stl_file_outside.substr(0, stl_file_outside.find(".stl"));

	
	std::cout << "Calculating file: " << stl_file_inside << "-" << stl_file_outside << std::endl;

	output.push_back(stl_file_inside + "-" + stl_file_outside + ";");

	Vertex* V1 = triangleMesh_Outside->getVertexAtIndex(0);
	Vertex* V2 = triangleMesh_Outside->getVertexAtIndex(1);
	Vertex* V3 = triangleMesh_Outside->getVertexAtIndex(2);

	float xCenter = (V1->getCoordinates()[0] + V2->getCoordinates()[0] + V3->getCoordinates()[0]) / 3;
	float yCenter = (V1->getCoordinates()[1] + V2->getCoordinates()[1] + V3->getCoordinates()[1]) / 3;
	float zCenter = (V1->getCoordinates()[2] + V2->getCoordinates()[2] + V3->getCoordinates()[2]) / 3;

	float direction[3] = { xCenter, yCenter, zCenter };

	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	//triangleMesh_Outside->rayTriangleIntersectOpenMP(direction, triangleMesh_Inside); // CPU version
	//triangleMesh_Outside->rayTriangleIntersect(direction, triangleMesh_Inside); // CPU version

	rayTriangle_BlockPerOrigin(direction, triangleMesh_Inside, triangleMesh_Outside);
	rayTriangle_ThreadPerOrigin(direction, triangleMesh_Inside, triangleMesh_Outside);
	rayTriangle_ThreadPerTriangle(direction, triangleMesh_Inside, triangleMesh_Outside);

	//triangleMesh_Outside->triangleTriangleIntersect(triangleMesh_Inside);

	TriangleTriangle_ThreadPerInnerTriangle(triangleMesh_Inside, triangleMesh_Outside); // GPU version
	TriangleTriangle_BlockPerInnerTriangle(triangleMesh_Inside, triangleMesh_Outside); // GPU version
	TriangleTriangle_ThreadPerOuterTriangle(triangleMesh_Inside, triangleMesh_Outside); // GPU version

	writeResultsToFile(output);

	std::cout << "Press Enter to quit program!" << std::endl;
	std::cin.get();
	std::cin.get();
	return 0;
}

void rayTriangle_BlockPerOrigin(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement
	startGPU<<<1,1>>>();
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();

	start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	float3* insideOrigins = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideOrigins;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideOrigins, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideOrigins, insideOrigins, sizeInsideVertices, hipMemcpyHostToDevice));
	
	float* cudaDir;
	handleCudaError(hipMalloc((void**)&cudaDir, 3 * sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3 * sizeof(float), hipMemcpyHostToDevice));

	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = outerMesh->getNumberOfVertices() * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	int totalIntersections = 0;

	hipEventRecord(start_event);
	intersect_triangleGPU_BlockPerOrigin<<<numberOfInsideVertices,128>>>(cudaInsideOrigins, cudaDir, cudaOutsideTriangles, cudaOutsideVertices, numberOfOutsideTriangles, cudaInside);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);
	
	hipFree(cudaInsideOrigins);
	hipFree(cudaDir);
	hipFree(cudaInside);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	
	hipHostFree(insideOrigins);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);

	std::string result;
	if (*inside) { result = "INSIDE"; }
	else { result = "OUTSIDE"; }
	output.push_back(std::to_string(milliseconds) + ";" + result + ";" + std::to_string((float)transferDuration/1000) + ";");

	delete inside;
}

void rayTriangle_ThreadPerOrigin(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	float3* insideOrigins = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideOrigins;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideOrigins, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideOrigins, insideOrigins, sizeInsideVertices, hipMemcpyHostToDevice));

	float* cudaDir;
	handleCudaError(hipMalloc((void**)&cudaDir, 3 * sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3 * sizeof(float), hipMemcpyHostToDevice));

	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = outerMesh->getNumberOfVertices() * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	
	int totalIntersections = 0;

	hipEventRecord(start_event);
	intersect_triangleGPU_ThreadPerOrigin<<<(numberOfInsideVertices+511)/512,512>>>(cudaInsideOrigins, cudaDir, cudaOutsideTriangles, cudaOutsideVertices, numberOfInsideVertices, numberOfOutsideTriangles, cudaInside);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);
	
	hipFree(cudaInsideOrigins);
	hipFree(cudaDir);
	hipFree(cudaInside);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	
	hipHostFree(insideOrigins);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	
	std::string result;
	if (*inside) { result = "INSIDE"; }
	else { result = "OUTSIDE"; }
	output.push_back(std::to_string(milliseconds) + ";" + result + ";" + std::to_string((float)transferDuration / 1000) + ";");

	delete inside;
}

void rayTriangle_ThreadPerTriangle(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool inside = true;
	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	float3* insideOrigins = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideOrigins;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideOrigins, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideOrigins, insideOrigins, sizeInsideVertices, hipMemcpyHostToDevice));

	float* cudaDir;
	handleCudaError(hipMalloc((void**)&cudaDir, 3 * sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3 * sizeof(float), hipMemcpyHostToDevice));

	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = outerMesh->getNumberOfVertices() * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));
	int* intersectionsPerOrigin = new int[numberOfInsideVertices];
	int* cudaIntersectionsPerOrigin;
	handleCudaError(hipMalloc((void**)&cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int)));

	
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	int totalIntersections = 0;

	hipEventRecord(start_event);
	intersect_triangleGPU_ThreadPerTriangle<<<(numberOfOutsideTriangles+127)/128,128>>>(cudaInsideOrigins, cudaDir, cudaOutsideTriangles, cudaOutsideVertices, numberOfInsideVertices, numberOfOutsideTriangles, cudaIntersectionsPerOrigin);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	handleCudaError(hipMemcpy(intersectionsPerOrigin, cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	int i = 0;
	while (i < numberOfInsideVertices && inside)
	{
		if (intersectionsPerOrigin[i] % 2 == 0) {
			inside = false;
		}
		i++;
	}

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	for (int i = 0; i < numberOfInsideVertices; i++)
	{
		totalIntersections += intersectionsPerOrigin[i];
	}

	hipFree(cudaInsideOrigins);
	hipFree(cudaDir);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaIntersectionsPerOrigin);
	
	hipHostFree(insideOrigins);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);

	delete intersectionsPerOrigin;

	std::string result;
	if (inside) { result = "INSIDE"; }
	else { result = "OUTSIDE"; }
	output.push_back(std::to_string(milliseconds) + ";" + result + ";" + std::to_string((float)transferDuration / 1000) + ";");
}

void TriangleTriangle_ThreadPerInnerTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfOutsideVertices = outerMesh->getNumberOfVertices();
	int numberOfInsideTriangles = innerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	/* Alloceren en kopi�ren hoekpunten binnenste mesh naar GPU*/
	float3* insideVertices = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideVertices;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideVertices, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideVertices, insideVertices, sizeInsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken binnenste mesh naar GPU*/
	int3* insideTriangles = innerMesh->getInt3ArrayTriangles();
	int3* cudaInsideTriangles;
	int sizeInsideTriangles = numberOfInsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaInsideTriangles, sizeInsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaInsideTriangles, insideTriangles, sizeInsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren hoekpunten buitenste mesh naar GPU*/
	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = numberOfOutsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken buitenste mesh naar GPU*/
	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	//TODO: extra lijst met outside mesh driehoekintervallen meegeven (ongesorteerd, omdat ik op dit moment geen idee heb waarom die gesorteerd zou moeten zijn)
	float2* outsideTriangleIntervals = outerMesh->getTriangleInterval();
	float2* cudaOutsideTriangleIntervals;
	int sizeOutsideIntervals = numberOfOutsideTriangles * sizeof(float2);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangleIntervals, sizeOutsideIntervals));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangleIntervals, outsideTriangleIntervals, sizeOutsideIntervals, hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	
	/* Als deze waarde > 0 ==> De binnenste mesh ligt niet volledig in de buitenste mesh*/
	int totalIntersections = 0;

	/****************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle met Broad Phase Collision Detection
	*****************************************************************************/
	/* Uitvoeren CUDA kernel*/
	hipEventRecord(start_event);
	triangle_triangle_GPU_BPCD_ThreadPerInnerTriangle<<<(numberOfInsideTriangles+511)/512,512>>>(cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, numberOfOutsideTriangles, cudaOutsideTriangleIntervals);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	/*******************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle zonder Broad Phase Collision Detection
	********************************************************************************/
	/* Uitvoeren CUDA kernel*/
	hipEventRecord(start_event);
	triangle_triangle_GPU_ThreadPerInnerTriangle<<<(numberOfInsideTriangles+511)/512,512>>>(cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, numberOfOutsideTriangles);
	hipEventRecord(stop_event);

	err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	std::string result;
	if (*inside) {
		result = "SNIJDEN NIET";
	}
	else {
		result = "SNIJDEN WEL";
	}
	output.push_back(std::to_string(milliseconds) + ";" + result + ";" + std::to_string((float)transferDuration / 1000) + ";");

	hipFree(cudaInsideTriangles);
	hipFree(cudaInsideVertices);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaInside);
	hipFree(cudaOutsideTriangleIntervals);
	//hipFree(cudaIntersectionsPerInsideTriangle);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	hipHostFree(insideTriangles);
	hipHostFree(insideVertices);
	hipHostFree(outsideTriangleIntervals);
	//delete intersectionsPerInsideTriangle;

	delete inside;
}

void TriangleTriangle_BlockPerInnerTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfOutsideVertices = outerMesh->getNumberOfVertices();
	int numberOfInsideTriangles = innerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	/* Alloceren en kopi�ren hoekpunten binnenste mesh naar GPU*/
	float3* insideVertices = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideVertices;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideVertices, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideVertices, insideVertices, sizeInsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken binnenste mesh naar GPU*/
	int3* insideTriangles = innerMesh->getInt3ArrayTriangles();
	int3* cudaInsideTriangles;
	int sizeInsideTriangles = numberOfInsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaInsideTriangles, sizeInsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaInsideTriangles, insideTriangles, sizeInsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren hoekpunten buitenste mesh naar GPU*/
	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = numberOfOutsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken buitenste mesh naar GPU*/
	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	//TODO: extra lijst met outside mesh driehoekintervallen meegeven (ongesorteerd, omdat ik op dit moment geen idee heb waarom die gesorteerd zou moeten zijn)
	float2* outsideTriangleIntervals = outerMesh->getTriangleInterval();
	float2* cudaOutsideTriangleIntervals;
	int sizeOutsideIntervals = numberOfOutsideTriangles * sizeof(float2);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangleIntervals, sizeOutsideIntervals));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangleIntervals, outsideTriangleIntervals, sizeOutsideIntervals, hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	/* Als deze waarde > 0 ==> De binnenste mesh ligt niet volledig in de buitenste mesh*/
	int totalIntersections = 0;

	/****************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle met Broad Phase Collision Detection
	*****************************************************************************/
	/* Uitvoeren CUDA kernel*/
	hipEventRecord(start_event);
	triangle_triangle_GPU_BPCD_BlockPerInnerTriangle<<<numberOfInsideTriangles,128>>>(cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, numberOfOutsideTriangles, cudaOutsideTriangleIntervals);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	/*******************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle zonder Broad Phase Collision Detection
	********************************************************************************/
	/* Uitvoeren CUDA kernel*/
	hipEventRecord(start_event);
	triangle_triangle_GPU_BlockPerInnerTriangle<<<numberOfInsideTriangles,128>>>(cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, numberOfOutsideTriangles);
	hipEventRecord(stop_event);

	err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	std::string result;
	if (*inside) {
		result = "SNIJDEN NIET";
	}
	else {
		result = "SNIJDEN WEL";
	}
	output.push_back(std::to_string(milliseconds) + ";" + result + ";" + std::to_string((float)transferDuration / 1000) + ";");

	hipFree(cudaInsideTriangles);
	hipFree(cudaInsideVertices);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaInside);
	hipFree(cudaOutsideTriangleIntervals);
	//hipFree(cudaIntersectionsPerInsideTriangle);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	hipHostFree(insideTriangles);
	hipHostFree(insideVertices);
	hipHostFree(outsideTriangleIntervals);
	//delete intersectionsPerInsideTriangle;

	delete inside;
}

void TriangleTriangle_ThreadPerOuterTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfOutsideVertices = outerMesh->getNumberOfVertices();
	int numberOfInsideTriangles = innerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	/* Alloceren en kopi�ren hoekpunten binnenste mesh naar GPU*/
	float3* insideVertices = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideVertices;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideVertices, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideVertices, insideVertices, sizeInsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken binnenste mesh naar GPU*/
	int3* insideTriangles = innerMesh->getInt3ArrayTriangles();
	int3* cudaInsideTriangles;
	int sizeInsideTriangles = numberOfInsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaInsideTriangles, sizeInsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaInsideTriangles, insideTriangles, sizeInsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren hoekpunten buitenste mesh naar GPU*/
	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = numberOfOutsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken buitenste mesh naar GPU*/
	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	//TODO: extra lijst met outside mesh driehoekintervallen meegeven (ongesorteerd, omdat ik op dit moment geen idee heb waarom die gesorteerd zou moeten zijn)
	float2* insideTriangleIntervals = innerMesh->getTriangleInterval();
	float2* cudaInsideTriangleIntervals;
	int sizeInsideIntervals = numberOfInsideTriangles * sizeof(float2);
	handleCudaError(hipMalloc((void**)&cudaInsideTriangleIntervals, sizeInsideIntervals));
	handleCudaError(hipMemcpyAsync(cudaInsideTriangleIntervals, insideTriangleIntervals, sizeInsideIntervals, hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	/* Als deze waarde > 0 ==> De binnenste mesh ligt niet volledig in de buitenste mesh*/
	int totalIntersections = 0;

	/****************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle met Broad Phase Collision Detection
	*****************************************************************************/
	/* Uitvoeren CUDA kernel*/
	hipEventRecord(start_event);
	triangle_triangle_GPU_BPCD_ThreadPerOuterTriangle<<<(numberOfOutsideTriangles+511)/512,512>>>(cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, numberOfOutsideTriangles, cudaInsideTriangleIntervals);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	/*******************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle zonder Broad Phase Collision Detection
	********************************************************************************/
	/* Uitvoeren CUDA kernel*/
	hipEventRecord(start_event);
	triangle_triangle_GPU_ThreadPerOuterTriangle<<<(numberOfOutsideTriangles+511)/512,512>>>(cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, numberOfOutsideTriangles);
	hipEventRecord(stop_event);

	err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	std::string result;
	if (*inside) {
		result = "SNIJDEN NIET";
	}
	else {
		result = "SNIJDEN WEL";
	}
	output.push_back(std::to_string(milliseconds) + ";" + result + ";" + std::to_string((float)transferDuration / 1000) + "\n");

	hipFree(cudaInsideTriangles);
	hipFree(cudaInsideVertices);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaInside);
	hipFree(cudaInsideTriangleIntervals);
	//hipFree(cudaIntersectionsPerInsideTriangle);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	hipHostFree(insideTriangles);
	hipHostFree(insideVertices);
	hipHostFree(insideTriangleIntervals);
	//delete intersectionsPerInsideTriangle;

	delete inside;
}

void handleCudaError(hipError_t cudaERR) {
	if (cudaERR != hipSuccess) {
		printf("CUDA ERROR : %s\n", hipGetErrorString(cudaERR));
	}
}

__global__ void startGPU() {
	printf("GPU ready!\n");
}

void writeResultsToFile(std::vector<std::string>& result)
{
	std::vector<std::string>::iterator itr;
	std::string path = "output.csv";
	std::ofstream ofs;
	ofs.open(path, std::ofstream::out | std::ofstream::app);
	for (itr = result.begin(); itr != result.end(); ++itr)
	{
		ofs << (*itr);
	}
}