#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <fstream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
//#include <memory> //needed for smart pointers

#include "Mesh.h"
#include "parse_stl.h"
#include "RayTriangleIntersect.cuh"
#include "TriangleTriangleIntersect.cuh"

void rayTriangle_BlockPerOrigin(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void rayTriangle_ThreadPerOrigin(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void rayTriangle_ThreadPerTriangle(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);

void TriangleTriangle_ThreadPerInnerTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void TriangleTriangle_BlockPerInnerTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void TriangleTriangle_ThreadPerOuterTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);

void handleCudaError(hipError_t cudaERR);
__global__ void startGPU();
void writeResultsToFile(std::vector<std::string>& result);

/* Console output wegschrijven naar file*/
std::vector<std::string> output;

int main(int argc, char* argv[]) {
	//output.push_back(";RT_v1(BPO);;RT_v2(TPO);;RT_v3(TPT);;TT_v1(TPIT);;TT_v2(BPIT);;TT_v3(TPOT);\n");

	std::string stl_file_inside;
	std::string stl_file_outside;
	std::cout << "Enter filename of inside mesh:" << std::endl;
	std::cin >> stl_file_inside;

	std::string delimiter = ".stl";
	std::string token = stl_file_inside.substr(28, stl_file_inside.find(delimiter));
	token = token.substr(0, token.find(delimiter));
	output.push_back(token + "-");

	std::cout << "Enter filename of outside mesh:" << std::endl;
	std::cin >> stl_file_outside;

	token = stl_file_outside.substr(28, stl_file_outside.find(delimiter));
	token = token.substr(0, token.find(delimiter));
	output.push_back(token + ";");

	if (argc == 2) {
		stl_file_inside = argv[1];
	}
	else if (argc > 2) {
		std::cout << "ERROR: Too many command line arguments" << std::endl;
	}

	auto t1 = std::chrono::high_resolution_clock::now(); //start time measurement

	//Only reads STL-file in binary format!!!
	std::cout << "Reading files:" << std::endl;
	std::unique_ptr<Mesh> triangleMesh_Inside = stl::parse_stl(stl_file_inside);
	std::unique_ptr<Mesh> triangleMesh_Outside = stl::parse_stl(stl_file_outside);

	auto t2 = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto time = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();
	std::cout << "Time = " << time << " milliseconds" << std::endl;

	std::cout << "STL HEADER = " << triangleMesh_Inside->getName() << std::endl;
	std::cout << "# triangles = " << triangleMesh_Inside->getNumberOfTriangles() << std::endl;
	std::cout << "# vertices = " << triangleMesh_Inside->getNumberOfVertices() << std::endl;

	std::cout << "STL HEADER = " << triangleMesh_Outside->getName() << std::endl;
	std::cout << "# triangles = " << triangleMesh_Outside->getNumberOfTriangles() << std::endl;
	std::cout << "# vertices = " << triangleMesh_Outside->getNumberOfVertices() << std::endl;

	Vertex* V1 = triangleMesh_Outside->getVertexAtIndex(0);
	Vertex* V2 = triangleMesh_Outside->getVertexAtIndex(1);
	Vertex* V3 = triangleMesh_Outside->getVertexAtIndex(2);

	float xCenter = (V1->getCoordinates()[0] + V2->getCoordinates()[0] + V3->getCoordinates()[0])/3;
	float yCenter = (V1->getCoordinates()[1] + V2->getCoordinates()[1] + V3->getCoordinates()[1])/3;
	float zCenter = (V1->getCoordinates()[2] + V2->getCoordinates()[2] + V3->getCoordinates()[2])/3;

	float direction[3] = { xCenter, yCenter, zCenter };

	std::cout << "direction = " << direction[0] << ", " << direction[1] << ", " << direction[2] << std::endl;

	//triangleMesh_Outside->rayTriangleIntersectOpenMP(direction, triangleMesh_Inside); // CPU version
	//triangleMesh_Outside->rayTriangleIntersect(direction, triangleMesh_Inside); // CPU version

	rayTriangle_BlockPerOrigin(direction, triangleMesh_Inside, triangleMesh_Outside);
	rayTriangle_ThreadPerOrigin(direction, triangleMesh_Inside, triangleMesh_Outside);
	rayTriangle_ThreadPerTriangle(direction, triangleMesh_Inside, triangleMesh_Outside);

	//triangleMesh_Outside->triangleTriangleIntersect(triangleMesh_Inside);

	TriangleTriangle_ThreadPerInnerTriangle(triangleMesh_Inside, triangleMesh_Outside); // GPU version
	TriangleTriangle_BlockPerInnerTriangle(triangleMesh_Inside, triangleMesh_Outside); // GPU version
	TriangleTriangle_ThreadPerOuterTriangle(triangleMesh_Inside, triangleMesh_Outside); // GPU version

	writeResultsToFile(output);

	std::cout << "Press Enter to quit program!" << std::endl;
	std::cin.get();
	std::cin.get();
	return 0;
}

void rayTriangle_BlockPerOrigin(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement
	startGPU<<<1,1>>>();
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "\t\t\tStartup time GPU = " << transferDuration << "ms" << std::endl;

	std::cout << "\t\t\tCalculating intersections! (GPU)" << std::endl;
	std::cout << "--- Data Transfer ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	float3* insideOrigins = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideOrigins;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideOrigins, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideOrigins, insideOrigins, sizeInsideVertices, hipMemcpyHostToDevice));
	
	float* cudaDir;
	handleCudaError(hipMalloc((void**)&cudaDir, 3 * sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3 * sizeof(float), hipMemcpyHostToDevice));

	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = outerMesh->getNumberOfVertices() * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	int totalIntersections = 0;

	std::cout << "--- End Data Transfer ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "\t\t\tTime Data Transfer = " << transferDuration << "ms" << std::endl;

	std::cout << "--- Calculating ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	int numberOfBlocks = numberOfInsideVertices;
	intersect_triangleGPU_BlockPerOrigin<<<numberOfBlocks,128>>>(cudaInsideOrigins, cudaDir, cudaOutsideTriangles, cudaOutsideVertices, numberOfOutsideTriangles, cudaInside);
	hipError_t err = hipGetLastError();
	handleCudaError(err);

	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));

	std::cout << "--- End Calculating ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto calculatingDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	std::cout << "\t\t\tTime Calculating = " << calculatingDuration << " microseconds" << std::endl;
	
	hipFree(cudaInsideOrigins);
	hipFree(cudaDir);
	hipFree(cudaInside);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	
	hipHostFree(insideOrigins);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);

	std::string result;
	std::cout << "totaal intersecties: " << totalIntersections << std::endl;
	if (*inside) { result = "INSIDE"; }
	else { result = "OUTSIDE"; }
	std::cout << result << std::endl;
	output.push_back(std::to_string(calculatingDuration) + ";" + result + ";");

	delete inside;
}

void rayTriangle_ThreadPerOrigin(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement
	startGPU<<<1,1>>>();
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "\t\t\tStartup time GPU = " << transferDuration << "ms" << std::endl;

	std::cout << "\t\t\tCalculating intersections! (GPU)" << std::endl;
	std::cout << "--- Data Transfer ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	float3* insideOrigins = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideOrigins;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideOrigins, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideOrigins, insideOrigins, sizeInsideVertices, hipMemcpyHostToDevice));

	float* cudaDir;
	handleCudaError(hipMalloc((void**)&cudaDir, 3 * sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3 * sizeof(float), hipMemcpyHostToDevice));

	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = outerMesh->getNumberOfVertices() * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	int totalIntersections = 0;

	std::cout << "--- End Data Transfer ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "\t\t\tTime Data Transfer = " << transferDuration << "ms" << std::endl;

	std::cout << "--- Calculating ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	int numberOfBlocks = ((int)((numberOfInsideVertices + 511) / 512));
	intersect_triangleGPU_ThreadPerOrigin<<<numberOfBlocks,512>>>(cudaInsideOrigins, cudaDir, cudaOutsideTriangles, cudaOutsideVertices, numberOfInsideVertices, numberOfOutsideTriangles, cudaInside);
	hipError_t err = hipGetLastError();
	handleCudaError(err);

	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));

	std::cout << "--- End Calculating ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto calculatingDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	std::cout << "\t\t\tTime Calculating = " << calculatingDuration << " microseconds" << std::endl;
	
	hipFree(cudaInsideOrigins);
	hipFree(cudaDir);
	hipFree(cudaInside);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	
	hipHostFree(insideOrigins);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	
	std::string result;
	std::cout << "totaal intersecties: " << totalIntersections << std::endl;
	if (*inside) { result = "INSIDE"; }
	else { result = "OUTSIDE"; }
	std::cout << result << std::endl;
	output.push_back(std::to_string(calculatingDuration) + ";" + result + ";");

	delete inside;
}

void rayTriangle_ThreadPerTriangle(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement
	startGPU<<<1,1>>>();
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "\t\t\tStartup time GPU = " << transferDuration << " milliseconds" << std::endl;

	std::cout << "\t\t\tCalculating intersections! (GPU)" << std::endl;
	std::cout << "--- Data Transfer ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool inside = true;
	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	float3* insideOrigins = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideOrigins;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideOrigins, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideOrigins, insideOrigins, sizeInsideVertices, hipMemcpyHostToDevice));

	float* cudaDir;
	handleCudaError(hipMalloc((void**)&cudaDir, 3 * sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3 * sizeof(float), hipMemcpyHostToDevice));

	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = outerMesh->getNumberOfVertices() * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));
	int* intersectionsPerOrigin = new int[numberOfInsideVertices];
	int* cudaIntersectionsPerOrigin;
	handleCudaError(hipMalloc((void**)&cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int)));

	int totalIntersections = 0;

	std::cout << "--- End Data Transfer ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "\t\t\tTime Data Transfer = " << transferDuration << " milliseconds" << std::endl;

	std::cout << "--- Calculating ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	int numberOfBlocks = ((int)((numberOfOutsideTriangles + 127) / 128));
	intersect_triangleGPU_ThreadPerTriangle<<<numberOfBlocks,128>>>(cudaInsideOrigins, cudaDir, cudaOutsideTriangles, cudaOutsideVertices, numberOfInsideVertices, numberOfOutsideTriangles, cudaIntersectionsPerOrigin);
	hipError_t err = hipGetLastError();
	handleCudaError(err);

	handleCudaError(hipMemcpy(intersectionsPerOrigin, cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int), hipMemcpyDeviceToHost));

	int i = 0;
	while (i < numberOfInsideVertices && inside)
	{
		if (intersectionsPerOrigin[i] % 2 == 0) {
			inside = false;
		}
		i++;
	}

	std::cout << "--- End Calculating ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto calculatingDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	std::cout << "\t\t\tTime Calculating = " << calculatingDuration << " microseconds" << std::endl;

	for (int i = 0; i < numberOfInsideVertices; i++)
	{
		totalIntersections += intersectionsPerOrigin[i];
	}

	hipFree(cudaInsideOrigins);
	hipFree(cudaDir);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaIntersectionsPerOrigin);
	
	hipHostFree(insideOrigins);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);

	delete intersectionsPerOrigin;

	std::string result;
	std::cout << "totaal intersecties: " << totalIntersections << std::endl;
	if (inside) { result = "INSIDE"; }
	else { result = "OUTSIDE"; }
	std::cout << result << std::endl;
	output.push_back(std::to_string(calculatingDuration) + ";" + result + ";");
}

void TriangleTriangle_ThreadPerInnerTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement
	startGPU<<<1,1>>>();
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "\t\t\tStartup time GPU = " << transferDuration << " milliseconds" << std::endl;

	std::cout << "\t\t\tCalculating intersections! (GPU)" << std::endl;
	std::cout << "--- Data Transfer ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfOutsideVertices = outerMesh->getNumberOfVertices();
	int numberOfInsideTriangles = innerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	/* Alloceren en kopi�ren hoekpunten binnenste mesh naar GPU*/
	float3* insideVertices = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideVertices;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideVertices, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideVertices, insideVertices, sizeInsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken binnenste mesh naar GPU*/
	int3* insideTriangles = innerMesh->getInt3ArrayTriangles();
	int3* cudaInsideTriangles;
	int sizeInsideTriangles = numberOfInsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaInsideTriangles, sizeInsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaInsideTriangles, insideTriangles, sizeInsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren hoekpunten buitenste mesh naar GPU*/
	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = numberOfOutsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken buitenste mesh naar GPU*/
	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	//TODO: extra lijst met outside mesh driehoekintervallen meegeven (ongesorteerd, omdat ik op dit moment geen idee heb waarom die gesorteerd zou moeten zijn)
	float2* outsideTriangleIntervals = outerMesh->getTriangleInterval();
	float2* cudaOutsideTriangleIntervals;
	int sizeOutsideIntervals = numberOfOutsideTriangles * sizeof(float2);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangleIntervals, sizeOutsideIntervals));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangleIntervals, outsideTriangleIntervals, sizeOutsideIntervals, hipMemcpyHostToDevice));

	/* Als deze waarde > 0 ==> De binnenste mesh ligt niet volledig in de buitenste mesh*/
	int totalIntersections = 0;

	std::cout << "--- End Data Transfer ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "\t\t\tTime Data Transfer = " << transferDuration << " milliseconds" << std::endl;


	/****************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle met Broad Phase Collision Detection
	*****************************************************************************/
	std::cout << "--- Calculating ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	/* Uitvoeren CUDA kernel*/
	int numberOfBlocks = ((int)((numberOfInsideTriangles + 511) / 512));

	triangle_triangle_GPU_BPCD_ThreadPerInnerTriangle<<<numberOfBlocks,512>>>(cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, numberOfOutsideTriangles, cudaOutsideTriangleIntervals);
	hipError_t err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));

	std::cout << "--- End Calculating ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto calculatingDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	std::cout << "\t\t\tTime Calculating (BPCD) = " << calculatingDuration << " microseconds" << std::endl;

	std::cout << "totaal intersecties: " << totalIntersections << std::endl;
	if (*inside) {
		std::cout << "SNIJDEN NIET" << std::endl;
	}
	else {
		std::cout << "SNIJDEN WEL" << std::endl;
	}

	/*******************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle zonder Broad Phase Collision Detection
	********************************************************************************/
	std::cout << "--- Calculating ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	/* Uitvoeren CUDA kernel*/
	numberOfBlocks = ((int)((numberOfInsideTriangles + 511) / 512));

	triangle_triangle_GPU_ThreadPerInnerTriangle<<<numberOfBlocks,512>>>(cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, numberOfOutsideTriangles);
	err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));

	std::cout << "--- End Calculating ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	calculatingDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	std::cout << "\t\t\tTime Calculating = " << calculatingDuration << " microseconds" << std::endl;

	std::string result;
	std::cout << "totaal intersecties: " << totalIntersections << std::endl;
	if (*inside) {
		result = "SNIJDEN NIET";
	}
	else {
		result = "SNIJDEN WEL";
	}
	std::cout << result << std::endl;
	output.push_back(std::to_string(calculatingDuration) + ";" + result + ";");

	hipFree(cudaInsideTriangles);
	hipFree(cudaInsideVertices);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaInside);
	hipFree(cudaOutsideTriangleIntervals);
	//hipFree(cudaIntersectionsPerInsideTriangle);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	hipHostFree(insideTriangles);
	hipHostFree(insideVertices);
	hipHostFree(outsideTriangleIntervals);
	//delete intersectionsPerInsideTriangle;

	delete inside;
}

void TriangleTriangle_BlockPerInnerTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement
	startGPU<<<1,1>>>();
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "\t\t\tStartup time GPU = " << transferDuration << " milliseconds" << std::endl;

	std::cout << "\t\t\tCalculating intersections! (GPU)" << std::endl;
	std::cout << "--- Data Transfer ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfOutsideVertices = outerMesh->getNumberOfVertices();
	int numberOfInsideTriangles = innerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	/* Alloceren en kopi�ren hoekpunten binnenste mesh naar GPU*/
	float3* insideVertices = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideVertices;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideVertices, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideVertices, insideVertices, sizeInsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken binnenste mesh naar GPU*/
	int3* insideTriangles = innerMesh->getInt3ArrayTriangles();
	int3* cudaInsideTriangles;
	int sizeInsideTriangles = numberOfInsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaInsideTriangles, sizeInsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaInsideTriangles, insideTriangles, sizeInsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren hoekpunten buitenste mesh naar GPU*/
	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = numberOfOutsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken buitenste mesh naar GPU*/
	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	//TODO: extra lijst met outside mesh driehoekintervallen meegeven (ongesorteerd, omdat ik op dit moment geen idee heb waarom die gesorteerd zou moeten zijn)
	float2* outsideTriangleIntervals = outerMesh->getTriangleInterval();
	float2* cudaOutsideTriangleIntervals;
	int sizeOutsideIntervals = numberOfOutsideTriangles * sizeof(float2);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangleIntervals, sizeOutsideIntervals));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangleIntervals, outsideTriangleIntervals, sizeOutsideIntervals, hipMemcpyHostToDevice));

	/* Als deze waarde > 0 ==> De binnenste mesh ligt niet volledig in de buitenste mesh*/
	int totalIntersections = 0;

	std::cout << "--- End Data Transfer ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "\t\t\tTime Data Transfer = " << transferDuration << " milliseconds" << std::endl;

	/****************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle met Broad Phase Collision Detection
	*****************************************************************************/
	std::cout << "--- Calculating ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	/* Uitvoeren CUDA kernel*/
	int numberOfBlocks = numberOfInsideTriangles;

	triangle_triangle_GPU_BPCD_BlockPerInnerTriangle<<<numberOfBlocks,128>>>(cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, numberOfOutsideTriangles, cudaOutsideTriangleIntervals);
	hipError_t err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));

	std::cout << "--- End Calculating ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto calculatingDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	std::cout << "\t\t\tTime Calculating (BPCD) = " << calculatingDuration << " microseconds" << std::endl;

	std::cout << "totaal intersecties: " << totalIntersections << std::endl;
	if (*inside) {
		std::cout << "SNIJDEN NIET" << std::endl;
	}
	else {
		std::cout << "SNIJDEN WEL" << std::endl;
	}

	/*******************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle zonder Broad Phase Collision Detection
	********************************************************************************/
	std::cout << "--- Calculating ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	/* Uitvoeren CUDA kernel*/
	numberOfBlocks = numberOfInsideTriangles;

	triangle_triangle_GPU_BlockPerInnerTriangle<<<numberOfBlocks,128>>>(cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, numberOfOutsideTriangles);
	err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));

	std::cout << "--- End Calculating ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	calculatingDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	std::cout << "\t\t\tTime Calculating = " << calculatingDuration << " microseconds" << std::endl;

	std::string result;
	std::cout << "totaal intersecties: " << totalIntersections << std::endl;
	if (*inside) {
		result = "SNIJDEN NIET";
	}
	else {
		result = "SNIJDEN WEL";
	}
	std::cout << result << std::endl;
	output.push_back(std::to_string(calculatingDuration) + ";" + result + ";");

	hipFree(cudaInsideTriangles);
	hipFree(cudaInsideVertices);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaInside);
	hipFree(cudaOutsideTriangleIntervals);
	//hipFree(cudaIntersectionsPerInsideTriangle);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	hipHostFree(insideTriangles);
	hipHostFree(insideVertices);
	hipHostFree(outsideTriangleIntervals);
	//delete intersectionsPerInsideTriangle;

	delete inside;
}

void TriangleTriangle_ThreadPerOuterTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement
	startGPU<<<1,1>>>();
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "\t\t\tStartup time GPU = " << transferDuration << " milliseconds" << std::endl;

	std::cout << "\t\t\tCalculating intersections! (GPU)" << std::endl;
	std::cout << "--- Data Transfer ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfOutsideVertices = outerMesh->getNumberOfVertices();
	int numberOfInsideTriangles = innerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	/* Alloceren en kopi�ren hoekpunten binnenste mesh naar GPU*/
	float3* insideVertices = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideVertices;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideVertices, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideVertices, insideVertices, sizeInsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken binnenste mesh naar GPU*/
	int3* insideTriangles = innerMesh->getInt3ArrayTriangles();
	int3* cudaInsideTriangles;
	int sizeInsideTriangles = numberOfInsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaInsideTriangles, sizeInsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaInsideTriangles, insideTriangles, sizeInsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren hoekpunten buitenste mesh naar GPU*/
	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = numberOfOutsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken buitenste mesh naar GPU*/
	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	//TODO: extra lijst met outside mesh driehoekintervallen meegeven (ongesorteerd, omdat ik op dit moment geen idee heb waarom die gesorteerd zou moeten zijn)
	float2* insideTriangleIntervals = innerMesh->getTriangleInterval();
	float2* cudaInsideTriangleIntervals;
	int sizeInsideIntervals = numberOfInsideTriangles * sizeof(float2);
	handleCudaError(hipMalloc((void**)&cudaInsideTriangleIntervals, sizeInsideIntervals));
	handleCudaError(hipMemcpyAsync(cudaInsideTriangleIntervals, insideTriangleIntervals, sizeInsideIntervals, hipMemcpyHostToDevice));

	/* Als deze waarde > 0 ==> De binnenste mesh ligt niet volledig in de buitenste mesh*/
	int totalIntersections = 0;

	std::cout << "--- End Data Transfer ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "\t\t\tTime Data Transfer = " << transferDuration << " milliseconds" << std::endl;

	/****************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle met Broad Phase Collision Detection
	*****************************************************************************/
	std::cout << "--- Calculating ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	/* Uitvoeren CUDA kernel*/
	int numberOfBlocks = ((int)((numberOfOutsideTriangles + 511) / 512));
	triangle_triangle_GPU_BPCD_ThreadPerOuterTriangle<<<numberOfBlocks,512>>>(cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, numberOfOutsideTriangles, cudaInsideTriangleIntervals);
	hipError_t err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));

	std::cout << "--- End Calculating ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto calculatingDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	std::cout << "\t\t\tTime Calculating (BPCD) = " << calculatingDuration << " microseconds" << std::endl;

	std::cout << "totaal intersecties: " << totalIntersections << std::endl;
	if (*inside) {
		std::cout << "SNIJDEN NIET" << std::endl;
	}
	else {
		std::cout << "SNIJDEN WEL" << std::endl;
	}

	/*******************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle zonder Broad Phase Collision Detection
	********************************************************************************/
	std::cout << "--- Calculating ---" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	/* Uitvoeren CUDA kernel*/
	numberOfBlocks = ((int)((numberOfOutsideTriangles + 511) / 512));
	triangle_triangle_GPU_ThreadPerOuterTriangle<<<numberOfBlocks,512>>>(cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, numberOfOutsideTriangles);
	err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));

	std::cout << "--- End Calculating ---" << std::endl;
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	calculatingDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	std::cout << "\t\t\tTime Calculating = " << calculatingDuration << " microseconds" << std::endl;

	std::string result;
	std::cout << "totaal intersecties: " << totalIntersections << std::endl;
	if (*inside) {
		result = "SNIJDEN NIET";
	}
	else {
		result = "SNIJDEN WEL";
	}
	std::cout << result << std::endl;
	output.push_back(std::to_string(calculatingDuration) + ";" + result + "\n");

	hipFree(cudaInsideTriangles);
	hipFree(cudaInsideVertices);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaInside);
	hipFree(cudaInsideTriangleIntervals);
	//hipFree(cudaIntersectionsPerInsideTriangle);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	hipHostFree(insideTriangles);
	hipHostFree(insideVertices);
	hipHostFree(insideTriangleIntervals);
	//delete intersectionsPerInsideTriangle;

	delete inside;
}

void handleCudaError(hipError_t cudaERR) {
	if (cudaERR != hipSuccess) {
		printf("CUDA ERROR : %s\n", hipGetErrorString(cudaERR));
	}
}

__global__ void startGPU() {
	printf("GPU ready!\n");
}

void writeResultsToFile(std::vector<std::string>& result)
{
	std::vector<std::string>::iterator itr;
	std::string path = "output.csv";
	std::ofstream ofs;
	ofs.open(path, std::ofstream::out | std::ofstream::app);
	for (itr = result.begin(); itr != result.end(); ++itr)
	{
		ofs << (*itr);
	}
}