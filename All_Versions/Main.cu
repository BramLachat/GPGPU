#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <fstream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
//#include <memory> //needed for smart pointers

#include "Mesh.h"
#include "parse_stl.h"
#include "RayTriangleIntersect.cuh"
#include "TriangleTriangleIntersect.cuh"

void rayTriangle_BlockPerOrigin(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void rayTriangle_ThreadPerOrigin(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void rayTriangle_ThreadPerTriangle(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void rayTriangle_BlockPerTriangle(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);

void TriangleTriangle_ThreadPerInnerTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void TriangleTriangle_ThreadPerInnerTriangle_BPCD(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);

void TriangleTriangle_BlockPerInnerTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void TriangleTriangle_BlockPerInnerTriangle_BPCD(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);

void TriangleTriangle_ThreadPerOuterTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void TriangleTriangle_ThreadPerOuterTriangle_BPCD(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);

void TriangleTriangle_BlockPerOuterTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);
void TriangleTriangle_BlockPerOuterTriangle_BPCD(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh);

void handleCudaError(hipError_t cudaERR);
__global__ void startGPU();
void writeResultsToFile(std::vector<std::string>& result);

/* Console output wegschrijven naar file*/
std::vector<std::string> output;

hipEvent_t start_event, stop_event;

int main(int argc, char* argv[]) {
	//output.push_back(";RT_v1(BPO)(ms);;;RT_v2(TPO)(ms);;;RT_v3(TPT)(ms);;;TT_v1(TPIT)(ms);;;TT_v2(BPIT)(ms);;;TT_v3(TPOT)(ms);;\n");

	std::string stl_file_inside;
	std::string stl_file_outside;
	std::cout << "Enter filename of inside mesh:" << std::endl;
	std::cin >> stl_file_inside;
	std::cout << "Enter filename of outside mesh:" << std::endl;
	std::cin >> stl_file_outside;


	std::string delimiter = "\\";

	//Only reads STL-file in binary format!!!
	std::cout << "Reading files:" << std::endl;
	std::unique_ptr<Mesh> triangleMesh_Inside = stl::parse_stl(stl_file_inside);
	std::unique_ptr<Mesh> triangleMesh_Outside = stl::parse_stl(stl_file_outside);

	size_t pos = 0;
	std::string token;
	while ((pos = stl_file_inside.find(delimiter)) != std::string::npos) {
		token = stl_file_inside.substr(0, pos);
		stl_file_inside.erase(0, pos + delimiter.length());
	}
	stl_file_inside = stl_file_inside.substr(0, stl_file_inside.find(".stl"));

	pos = 0;
	while ((pos = stl_file_outside.find(delimiter)) != std::string::npos) {
		token = stl_file_outside.substr(0, pos);
		stl_file_outside.erase(0, pos + delimiter.length());
	}
	stl_file_outside = stl_file_outside.substr(0, stl_file_outside.find(".stl"));


	std::cout << "Calculating file: " << stl_file_inside << "-" << stl_file_outside << std::endl;

	output.push_back(stl_file_inside + "-" + stl_file_outside + ";");

	Vertex* V1 = triangleMesh_Outside->getVertexAtIndex(0);
	Vertex* V2 = triangleMesh_Outside->getVertexAtIndex(1);
	Vertex* V3 = triangleMesh_Outside->getVertexAtIndex(2);

	float xCenter = (V1->getCoordinates()[0] + V2->getCoordinates()[0] + V3->getCoordinates()[0]) / 3;
	float yCenter = (V1->getCoordinates()[1] + V2->getCoordinates()[1] + V3->getCoordinates()[1]) / 3;
	float zCenter = (V1->getCoordinates()[2] + V2->getCoordinates()[2] + V3->getCoordinates()[2]) / 3;

	float direction[3] = { xCenter, yCenter, zCenter };

	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	//triangleMesh_Outside->rayTriangleIntersectOpenMP(direction, triangleMesh_Inside); // CPU version
	//triangleMesh_Outside->rayTriangleIntersect(direction, triangleMesh_Inside); // CPU version

	/*rayTriangle_BlockPerOrigin(direction, triangleMesh_Inside, triangleMesh_Outside);
	rayTriangle_ThreadPerOrigin(direction, triangleMesh_Inside, triangleMesh_Outside);
	rayTriangle_ThreadPerTriangle(direction, triangleMesh_Inside, triangleMesh_Outside);
	rayTriangle_BlockPerTriangle(direction, triangleMesh_Inside, triangleMesh_Outside);*/

	//triangleMesh_Outside->triangleTriangleIntersect(triangleMesh_Inside); // CPU

				/*int threads = 16; 
				while (threads < 513) {
					TriangleTriangle_BlockPerInnerTriangle(triangleMesh_Inside, triangleMesh_Outside); // GPU version
					threads = threads * 2;
				}*/

	//TriangleTriangle_ThreadPerInnerTriangle(triangleMesh_Inside, triangleMesh_Outside); // GPU version
	TriangleTriangle_ThreadPerInnerTriangle_BPCD(triangleMesh_Inside, triangleMesh_Outside); // GPU version

	//TriangleTriangle_BlockPerInnerTriangle(triangleMesh_Inside, triangleMesh_Outside); // GPU version
	TriangleTriangle_BlockPerInnerTriangle_BPCD(triangleMesh_Inside, triangleMesh_Outside); // GPU version

	//TriangleTriangle_ThreadPerOuterTriangle(triangleMesh_Inside, triangleMesh_Outside); // GPU version
	TriangleTriangle_ThreadPerOuterTriangle_BPCD(triangleMesh_Inside, triangleMesh_Outside); // GPU version

	//TriangleTriangle_BlockPerOuterTriangle(triangleMesh_Inside, triangleMesh_Outside); // GPU version
	TriangleTriangle_BlockPerOuterTriangle_BPCD(triangleMesh_Inside, triangleMesh_Outside); // GPU version

	output.push_back("\n");

	writeResultsToFile(output);

	std::cout << "Press Enter to quit program!" << std::endl;
	std::cin.get();
	std::cin.get();
	return 0;
}

void rayTriangle_BlockPerOrigin(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement
	startGPU<<<1,1>>>();
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();

	std::cout << "Transfering data from cpu to gpu!" << std::endl;
	start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	float3* insideOrigins = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideOrigins;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideOrigins, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideOrigins, insideOrigins, sizeInsideVertices, hipMemcpyHostToDevice));

	float* cudaDir;
	handleCudaError(hipMalloc((void**)&cudaDir, 3 * sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3 * sizeof(float), hipMemcpyHostToDevice));

	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = outerMesh->getNumberOfVertices() * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	end = std::chrono::high_resolution_clock::now(); //stop time measurement
	transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	int totalIntersections = 0;
	std::cout << "Kernel execution: rayTriangle_BlockPerOrigin" << std::endl;

	hipEventRecord(start_event);
	intersect_triangleGPU_BlockPerOrigin << <numberOfInsideVertices, 128 >> > (cudaInsideOrigins, cudaDir, cudaOutsideTriangles, cudaOutsideVertices, numberOfOutsideTriangles, cudaInside);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	hipFree(cudaInsideOrigins);
	hipFree(cudaDir);
	hipFree(cudaInside);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);

	hipHostFree(insideOrigins);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);

	std::string result;
	if (*inside) { result = "INSIDE"; }
	else { result = "OUTSIDE"; }
	std::cout << result << std::endl;
	output.push_back(std::to_string(milliseconds) + ";" + result + ";" + std::to_string((float)transferDuration / 1000) + ";");

	delete inside;
}

void rayTriangle_ThreadPerOrigin(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	std::cout << "Transfering data from cpu to gpu!" << std::endl;
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	float3* insideOrigins = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideOrigins;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideOrigins, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideOrigins, insideOrigins, sizeInsideVertices, hipMemcpyHostToDevice));

	float* cudaDir;
	handleCudaError(hipMalloc((void**)&cudaDir, 3 * sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3 * sizeof(float), hipMemcpyHostToDevice));

	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = outerMesh->getNumberOfVertices() * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	int totalIntersections = 0;
	std::cout << "Kernel execution: rayTriangle_ThreadPerOrigin" << std::endl;

	hipEventRecord(start_event);
	intersect_triangleGPU_ThreadPerOrigin << <(numberOfInsideVertices + 511) / 512, 512 >> > (cudaInsideOrigins, cudaDir, cudaOutsideTriangles, cudaOutsideVertices, numberOfInsideVertices, numberOfOutsideTriangles, cudaInside);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	hipFree(cudaInsideOrigins);
	hipFree(cudaDir);
	hipFree(cudaInside);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);

	hipHostFree(insideOrigins);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);

	std::string result;
	if (*inside) { result = "INSIDE"; }
	else { result = "OUTSIDE"; }
	std::cout << result << std::endl;
	output.push_back(std::to_string(milliseconds) + ";" + result + ";" + std::to_string((float)transferDuration / 1000) + ";");

	delete inside;
}

void rayTriangle_ThreadPerTriangle(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	std::cout << "Transfering data from cpu to gpu!" << std::endl;
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool inside = true;
	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	float3* insideOrigins = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideOrigins;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideOrigins, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideOrigins, insideOrigins, sizeInsideVertices, hipMemcpyHostToDevice));

	float* cudaDir;
	handleCudaError(hipMalloc((void**)&cudaDir, 3 * sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3 * sizeof(float), hipMemcpyHostToDevice));

	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = outerMesh->getNumberOfVertices() * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));
	int* intersectionsPerOrigin = new int[numberOfInsideVertices];
	int* cudaIntersectionsPerOrigin;
	handleCudaError(hipMalloc((void**)&cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int)));


	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	int totalIntersections = 0;
	std::cout << "Kernel execution: rayTriangle_ThreadPerTriangle" << std::endl;

	hipEventRecord(start_event);
	intersect_triangleGPU_ThreadPerTriangle << <(numberOfOutsideTriangles + 127) / 128, 128 >> > (cudaInsideOrigins, cudaDir, cudaOutsideTriangles, cudaOutsideVertices, numberOfInsideVertices, numberOfOutsideTriangles, cudaIntersectionsPerOrigin);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	handleCudaError(hipMemcpy(intersectionsPerOrigin, cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	int i = 0;
	while (i < numberOfInsideVertices && inside)
	{
		if (intersectionsPerOrigin[i] % 2 == 0) {
			inside = false;
		}
		i++;
	}

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	for (int i = 0; i < numberOfInsideVertices; i++)
	{
		totalIntersections += intersectionsPerOrigin[i];
	}

	hipFree(cudaInsideOrigins);
	hipFree(cudaDir);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaIntersectionsPerOrigin);

	hipHostFree(insideOrigins);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);

	delete intersectionsPerOrigin;

	std::string result;
	if (inside) { result = "INSIDE"; }
	else { result = "OUTSIDE"; }
	std::cout << result << std::endl;
	output.push_back(std::to_string(milliseconds) + ";" + result + ";" + std::to_string((float)transferDuration / 1000) + ";");
}

void rayTriangle_BlockPerTriangle(float dir[3], std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	std::cout << "Transfering data from cpu to gpu!" << std::endl;
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	float3* insideOrigins = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideOrigins;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideOrigins, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideOrigins, insideOrigins, sizeInsideVertices, hipMemcpyHostToDevice));

	float* cudaDir;
	handleCudaError(hipMalloc((void**)&cudaDir, 3 * sizeof(float)));
	handleCudaError(hipMemcpy(cudaDir, dir, 3 * sizeof(float), hipMemcpyHostToDevice));

	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = outerMesh->getNumberOfVertices() * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	int* intersectionsPerOrigin = new int[numberOfInsideVertices];
	int* cudaIntersectionsPerOrigin;
	handleCudaError(hipMalloc((void**)&cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int)));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	int totalIntersections = 0;
	std::cout << "Kernel execution: rayTriangle_BlockPerTriangle" << std::endl;

	hipEventRecord(start_event);
	intersect_triangleGPU_BlockPerTriangle << <numberOfOutsideTriangles, 128 >> > (cudaInsideOrigins, cudaDir, cudaOutsideTriangles, cudaOutsideVertices, numberOfInsideVertices, cudaIntersectionsPerOrigin);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	handleCudaError(hipMemcpy(intersectionsPerOrigin, cudaIntersectionsPerOrigin, numberOfInsideVertices * sizeof(int), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	int i = 0;
	while (i < numberOfInsideVertices && inside)
	{
		if (intersectionsPerOrigin[i] % 2 == 0) {
			inside = false;
		}
		i++;
	}

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	for (int i = 0; i < numberOfInsideVertices; i++)
	{
		totalIntersections += intersectionsPerOrigin[i];
	}

	hipFree(cudaInsideOrigins);
	hipFree(cudaDir);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaIntersectionsPerOrigin);

	hipHostFree(insideOrigins);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);

	delete intersectionsPerOrigin;

	std::string result;
	if (*inside) { result = "INSIDE"; }
	else { result = "OUTSIDE"; }
	std::cout << result << std::endl;
	output.push_back(std::to_string(milliseconds) + ";" + result + ";" + std::to_string((float)transferDuration / 1000) + ";");

	delete inside;
}

void TriangleTriangle_ThreadPerInnerTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	std::cout << "Transfering data from cpu to gpu!" << std::endl;
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfOutsideVertices = outerMesh->getNumberOfVertices();
	int numberOfInsideTriangles = innerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	/* Alloceren en kopi�ren hoekpunten binnenste mesh naar GPU*/
	float3* insideVertices = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideVertices;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideVertices, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideVertices, insideVertices, sizeInsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken binnenste mesh naar GPU*/
	int3* insideTriangles = innerMesh->getInt3ArrayTriangles();
	int3* cudaInsideTriangles;
	int sizeInsideTriangles = numberOfInsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaInsideTriangles, sizeInsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaInsideTriangles, insideTriangles, sizeInsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren hoekpunten buitenste mesh naar GPU*/
	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = numberOfOutsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken buitenste mesh naar GPU*/
	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	/* Als deze waarde > 0 ==> De binnenste mesh ligt niet volledig in de buitenste mesh*/
	int totalIntersections = 0;
	std::cout << "Kernel execution: TriangleTriangle_ThreadPerInnerTriangle" << std::endl;

	/*******************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle zonder Broad Phase Collision Detection
	********************************************************************************/
	/* Uitvoeren CUDA kernel*/
	hipEventRecord(start_event);
	triangle_triangle_GPU_ThreadPerInnerTriangle << <(numberOfInsideTriangles + 255) / 256, 256 >> > (cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, numberOfOutsideTriangles);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	std::string result;
	if (*inside) {
		result = "SNIJDEN NIET";
	}
	else {
		result = "SNIJDEN WEL";
	}
	std::cout << result << std::endl;
	output.push_back(std::to_string(milliseconds) + ";" + result + ";" + std::to_string((float)transferDuration / 1000) + ";");

	hipFree(cudaInsideTriangles);
	hipFree(cudaInsideVertices);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaInside);
	//hipFree(cudaIntersectionsPerInsideTriangle);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	hipHostFree(insideTriangles);
	hipHostFree(insideVertices);
	//delete intersectionsPerInsideTriangle;

	delete inside;
}

void TriangleTriangle_ThreadPerInnerTriangle_BPCD(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	std::cout << "Transfering data from cpu to gpu!" << std::endl;
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfOutsideVertices = outerMesh->getNumberOfVertices();
	int numberOfInsideTriangles = innerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	/* Alloceren en kopi�ren hoekpunten binnenste mesh naar GPU*/
	float3* insideVertices = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideVertices;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideVertices, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideVertices, insideVertices, sizeInsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken binnenste mesh naar GPU*/
	int3* insideTriangles = innerMesh->getInt3ArrayTriangles();
	int3* cudaInsideTriangles;
	int sizeInsideTriangles = numberOfInsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaInsideTriangles, sizeInsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaInsideTriangles, insideTriangles, sizeInsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren hoekpunten buitenste mesh naar GPU*/
	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = numberOfOutsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken buitenste mesh naar GPU*/
	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren voor mogelijke driehoeken die kunnen snijden uit BPCD op GPU*/
	int* intersectingTriangles;
	int sizeIntersectingTriangles = numberOfInsideTriangles * 10 * sizeof(int);
	handleCudaError(hipMalloc((void**)&intersectingTriangles, sizeIntersectingTriangles));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	/* Als deze waarde > 0 ==> De binnenste mesh ligt niet volledig in de buitenste mesh*/
	int totalIntersections = 0;
	std::cout << "Kernel execution: TriangleTriangle_ThreadPerInnerTriangle_BPCD" << std::endl;

	/*******************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle met Broad Phase Collision Detection
	********************************************************************************/
	/* Uitvoeren CUDA kernel*/
	hipEventRecord(start_event);
	triangle_triangle_GPU_BPCD_1_ThreadPerInnerTriangle<<<(numberOfInsideTriangles + 255) / 256, 256>>>(cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, numberOfInsideTriangles, numberOfOutsideTriangles, intersectingTriangles);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);
	hipEventSynchronize(stop_event);
	float milliseconds_1 = 0;
	hipEventElapsedTime(&milliseconds_1, start_event, stop_event);

	hipEventRecord(start_event);
	triangle_triangle_GPU_BPCD_2_ThreadPerInnerTriangle << <(numberOfInsideTriangles + 255) / 256, 256 >> > (cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, intersectingTriangles);
	hipEventRecord(stop_event);

	err = hipGetLastError();
	handleCudaError(err);
	
	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);
	float milliseconds_2 = 0;
	hipEventElapsedTime(&milliseconds_2, start_event, stop_event);

	std::string result;
	if (*inside) {
		result = "SNIJDEN NIET";
	}
	else {
		result = "SNIJDEN WEL";
	}
	std::cout << result << std::endl;
	output.push_back(std::to_string(milliseconds_1) + ";" + std::to_string(milliseconds_2) + ";" + std::to_string(milliseconds_1 + milliseconds_2) + ";" + result + ";" + std::to_string((float)transferDuration / 1000) + ";");

	hipFree(cudaInsideTriangles);
	hipFree(cudaInsideVertices);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaInside);
	hipFree(intersectingTriangles);
	//hipFree(cudaIntersectionsPerInsideTriangle);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	hipHostFree(insideTriangles);
	hipHostFree(insideVertices);
	//delete intersectionsPerInsideTriangle;

	delete inside;
}

void TriangleTriangle_BlockPerInnerTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	std::cout << "Transfering data from cpu to gpu!" << std::endl;
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfOutsideVertices = outerMesh->getNumberOfVertices();
	int numberOfInsideTriangles = innerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	/* Alloceren en kopi�ren hoekpunten binnenste mesh naar GPU*/
	float3* insideVertices = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideVertices;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideVertices, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideVertices, insideVertices, sizeInsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken binnenste mesh naar GPU*/
	int3* insideTriangles = innerMesh->getInt3ArrayTriangles();
	int3* cudaInsideTriangles;
	int sizeInsideTriangles = numberOfInsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaInsideTriangles, sizeInsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaInsideTriangles, insideTriangles, sizeInsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren hoekpunten buitenste mesh naar GPU*/
	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = numberOfOutsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken buitenste mesh naar GPU*/
	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	/* Als deze waarde > 0 ==> De binnenste mesh ligt niet volledig in de buitenste mesh*/
	int totalIntersections = 0;
	std::cout << "Kernel execution: TriangleTriangle_BlockPerInnerTriangle" << std::endl;

	/*******************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle zonder Broad Phase Collision Detection
	********************************************************************************/
	/* Uitvoeren CUDA kernel*/
	hipEventRecord(start_event);
	triangle_triangle_GPU_BlockPerInnerTriangle << <numberOfInsideTriangles, 128 >> > (cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, numberOfOutsideTriangles);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	std::string result;
	if (*inside) {
		result = "SNIJDEN NIET";
	}
	else {
		result = "SNIJDEN WEL";
	}
	std::cout << result << std::endl;
	output.push_back(std::to_string(milliseconds) + ";" + result + ";" + std::to_string((float)transferDuration / 1000) + ";");

	hipFree(cudaInsideTriangles);
	hipFree(cudaInsideVertices);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaInside);
	//hipFree(cudaIntersectionsPerInsideTriangle);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	hipHostFree(insideTriangles);
	hipHostFree(insideVertices);
	//delete intersectionsPerInsideTriangle;

	delete inside;
}

void TriangleTriangle_BlockPerInnerTriangle_BPCD(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	std::cout << "Transfering data from cpu to gpu!" << std::endl;
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfOutsideVertices = outerMesh->getNumberOfVertices();
	int numberOfInsideTriangles = innerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	/* Alloceren en kopi�ren hoekpunten binnenste mesh naar GPU*/
	float3* insideVertices = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideVertices;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideVertices, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideVertices, insideVertices, sizeInsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken binnenste mesh naar GPU*/
	int3* insideTriangles = innerMesh->getInt3ArrayTriangles();
	int3* cudaInsideTriangles;
	int sizeInsideTriangles = numberOfInsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaInsideTriangles, sizeInsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaInsideTriangles, insideTriangles, sizeInsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren hoekpunten buitenste mesh naar GPU*/
	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = numberOfOutsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken buitenste mesh naar GPU*/
	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren voor mogelijke driehoeken die kunnen snijden uit BPCD op GPU*/
	int* intersectingTriangles;
	int sizeIntersectingTriangles = numberOfInsideTriangles * 10 * sizeof(int);
	handleCudaError(hipMalloc((void**)&intersectingTriangles, sizeIntersectingTriangles));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	/* Als deze waarde > 0 ==> De binnenste mesh ligt niet volledig in de buitenste mesh*/
	int totalIntersections = 0;
	std::cout << "Kernel execution: TriangleTriangle_BlockPerInnerTriangle_BPCD" << std::endl;

	/*******************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle met Broad Phase Collision Detection
	********************************************************************************/
	/* Uitvoeren CUDA kernel*/
	hipEventRecord(start_event);
	triangle_triangle_GPU_BPCD_1_ThreadPerInnerTriangle << <(numberOfInsideTriangles + 255) / 256, 256 >> > (cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, numberOfInsideTriangles, numberOfOutsideTriangles, intersectingTriangles);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);
	hipEventSynchronize(stop_event);
	float milliseconds_1 = 0;
	hipEventElapsedTime(&milliseconds_1, start_event, stop_event);

	hipEventRecord(start_event);
	triangle_triangle_GPU_BPCD_2_BlockPerInnerTriangle << <numberOfInsideTriangles, 128 >> > (cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, intersectingTriangles);
	hipEventRecord(stop_event);

	err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);
	float milliseconds_2 = 0;
	hipEventElapsedTime(&milliseconds_2, start_event, stop_event);

	std::string result;
	if (*inside) {
		result = "SNIJDEN NIET";
	}
	else {
		result = "SNIJDEN WEL";
	}
	std::cout << result << std::endl;
	output.push_back(std::to_string(milliseconds_1) + ";" + std::to_string(milliseconds_2) + ";" + std::to_string(milliseconds_1 + milliseconds_2) + ";" + result + ";" + std::to_string((float)transferDuration / 1000) + ";");

	hipFree(cudaInsideTriangles);
	hipFree(cudaInsideVertices);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaInside);
	hipFree(intersectingTriangles);
	//hipFree(cudaIntersectionsPerInsideTriangle);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	hipHostFree(insideTriangles);
	hipHostFree(insideVertices);
	//delete intersectionsPerInsideTriangle;

	delete inside;
}

void TriangleTriangle_ThreadPerOuterTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	std::cout << "Transfering data from cpu to gpu!" << std::endl;
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfOutsideVertices = outerMesh->getNumberOfVertices();
	int numberOfInsideTriangles = innerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	/* Alloceren en kopi�ren hoekpunten binnenste mesh naar GPU*/
	float3* insideVertices = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideVertices;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideVertices, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideVertices, insideVertices, sizeInsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken binnenste mesh naar GPU*/
	int3* insideTriangles = innerMesh->getInt3ArrayTriangles();
	int3* cudaInsideTriangles;
	int sizeInsideTriangles = numberOfInsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaInsideTriangles, sizeInsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaInsideTriangles, insideTriangles, sizeInsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren hoekpunten buitenste mesh naar GPU*/
	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = numberOfOutsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken buitenste mesh naar GPU*/
	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	/* Als deze waarde > 0 ==> De binnenste mesh ligt niet volledig in de buitenste mesh*/
	int totalIntersections = 0;
	std::cout << "Kernel execution: TriangleTriangle_ThreadPerOuterTriangle" << std::endl;

	/*******************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle zonder Broad Phase Collision Detection
	********************************************************************************/
	/* Uitvoeren CUDA kernel*/
	hipEventRecord(start_event);
	triangle_triangle_GPU_ThreadPerOuterTriangle << <(numberOfOutsideTriangles + 511) / 512, 512 >> > (cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, numberOfOutsideTriangles);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	std::string result;
	if (*inside) {
		result = "SNIJDEN NIET";
	}
	else {
		result = "SNIJDEN WEL";
	}
	std::cout << result << std::endl;
	output.push_back(std::to_string(milliseconds) + ";" + result + ";" + std::to_string((float)transferDuration / 1000) + ";");

	hipFree(cudaInsideTriangles);
	hipFree(cudaInsideVertices);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaInside);
	//hipFree(cudaIntersectionsPerInsideTriangle);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	hipHostFree(insideTriangles);
	hipHostFree(insideVertices);
	//delete intersectionsPerInsideTriangle;

	delete inside;
}

void TriangleTriangle_ThreadPerOuterTriangle_BPCD(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh) 
{
	std::cout << "Transfering data from cpu to gpu!" << std::endl;
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfOutsideVertices = outerMesh->getNumberOfVertices();
	int numberOfInsideTriangles = innerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	/* Alloceren en kopi�ren hoekpunten binnenste mesh naar GPU*/
	float3* insideVertices = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideVertices;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideVertices, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideVertices, insideVertices, sizeInsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken binnenste mesh naar GPU*/
	int3* insideTriangles = innerMesh->getInt3ArrayTriangles();
	int3* cudaInsideTriangles;
	int sizeInsideTriangles = numberOfInsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaInsideTriangles, sizeInsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaInsideTriangles, insideTriangles, sizeInsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren hoekpunten buitenste mesh naar GPU*/
	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = numberOfOutsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken buitenste mesh naar GPU*/
	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren voor mogelijke driehoeken die kunnen snijden uit BPCD op GPU*/
	int* intersectingTriangles;
	int sizeIntersectingTriangles = numberOfOutsideTriangles * 10 * sizeof(int);
	handleCudaError(hipMalloc((void**)&intersectingTriangles, sizeIntersectingTriangles));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	/* Als deze waarde > 0 ==> De binnenste mesh ligt niet volledig in de buitenste mesh*/
	int totalIntersections = 0;
	std::cout << "Kernel execution: TriangleTriangle_ThreadPerOuterTriangle_BPCD" << std::endl;

	/*******************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle met Broad Phase Collision Detection
	********************************************************************************/
	/* Uitvoeren CUDA kernel*/
	hipEventRecord(start_event);
	triangle_triangle_GPU_BPCD_1_ThreadPerOuterTriangle << <(numberOfOutsideTriangles + 255) / 256, 256 >> > (cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, numberOfInsideTriangles, numberOfOutsideTriangles, intersectingTriangles);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);
	hipEventSynchronize(stop_event);
	float milliseconds_1 = 0;
	hipEventElapsedTime(&milliseconds_1, start_event, stop_event);

	hipEventRecord(start_event);
	triangle_triangle_GPU_BPCD_2_ThreadPerOuterTriangle << <(numberOfOutsideTriangles + 255) / 256, 256 >> > (cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfOutsideTriangles, intersectingTriangles);
	hipEventRecord(stop_event);

	err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);
	float milliseconds_2 = 0;
	hipEventElapsedTime(&milliseconds_2, start_event, stop_event);

	std::string result;
	if (*inside) {
		result = "SNIJDEN NIET";
	}
	else {
		result = "SNIJDEN WEL";
	}
	std::cout << result << std::endl;
	output.push_back(std::to_string(milliseconds_1) + ";" + std::to_string(milliseconds_2) + ";" + std::to_string(milliseconds_1 + milliseconds_2) + ";" + result + ";" + std::to_string((float)transferDuration / 1000) + ";");


	hipFree(cudaInsideTriangles);
	hipFree(cudaInsideVertices);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaInside);
	hipFree(intersectingTriangles);
	//hipFree(cudaIntersectionsPerInsideTriangle);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	hipHostFree(insideTriangles);
	hipHostFree(insideVertices);
	//delete intersectionsPerInsideTriangle;

	delete inside;
}

void TriangleTriangle_BlockPerOuterTriangle(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh)
{
	std::cout << "Transfering data from cpu to gpu!" << std::endl;
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfOutsideVertices = outerMesh->getNumberOfVertices();
	int numberOfInsideTriangles = innerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	/* Alloceren en kopi�ren hoekpunten binnenste mesh naar GPU*/
	float3* insideVertices = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideVertices;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideVertices, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideVertices, insideVertices, sizeInsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken binnenste mesh naar GPU*/
	int3* insideTriangles = innerMesh->getInt3ArrayTriangles();
	int3* cudaInsideTriangles;
	int sizeInsideTriangles = numberOfInsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaInsideTriangles, sizeInsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaInsideTriangles, insideTriangles, sizeInsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren hoekpunten buitenste mesh naar GPU*/
	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = numberOfOutsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken buitenste mesh naar GPU*/
	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	/* Als deze waarde > 0 ==> De binnenste mesh ligt niet volledig in de buitenste mesh*/
	int totalIntersections = 0;
	std::cout << "Kernel execution: TriangleTriangle_BlockPerOuterTriangle" << std::endl;

	/*******************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle zonder Broad Phase Collision Detection
	********************************************************************************/
	/* Uitvoeren CUDA kernel*/
	hipEventRecord(start_event);
	triangle_triangle_GPU_BlockPerOuterTriangle << <numberOfOutsideTriangles, 128 >> > (cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, numberOfInsideTriangles, numberOfOutsideTriangles);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);

	std::string result;
	if (*inside) {
		result = "SNIJDEN NIET";
	}
	else {
		result = "SNIJDEN WEL";
	}
	std::cout << result << std::endl;
	output.push_back(std::to_string(milliseconds) + ";" + result + ";" + std::to_string((float)transferDuration / 1000) + ";");

	hipFree(cudaInsideTriangles);
	hipFree(cudaInsideVertices);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaInside);
	//hipFree(cudaIntersectionsPerInsideTriangle);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	hipHostFree(insideTriangles);
	hipHostFree(insideVertices);
	//delete intersectionsPerInsideTriangle;

	delete inside;
}

void TriangleTriangle_BlockPerOuterTriangle_BPCD(std::unique_ptr<Mesh>& innerMesh, std::unique_ptr<Mesh>& outerMesh) 
{
	std::cout << "Transfering data from cpu to gpu!" << std::endl;
	auto start = std::chrono::high_resolution_clock::now(); //start time measurement

	bool* inside = new bool;
	*inside = true;
	bool* cudaInside;
	handleCudaError(hipMalloc((void**)&cudaInside, sizeof(bool)));
	handleCudaError(hipMemcpy(cudaInside, inside, sizeof(bool), hipMemcpyHostToDevice));

	int numberOfOutsideTriangles = outerMesh->getNumberOfTriangles();
	int numberOfOutsideVertices = outerMesh->getNumberOfVertices();
	int numberOfInsideTriangles = innerMesh->getNumberOfTriangles();
	int numberOfInsideVertices = innerMesh->getNumberOfVertices();

	/* Alloceren en kopi�ren hoekpunten binnenste mesh naar GPU*/
	float3* insideVertices = innerMesh->getFloat3ArrayVertices();
	float3* cudaInsideVertices;
	int sizeInsideVertices = numberOfInsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaInsideVertices, sizeInsideVertices));
	handleCudaError(hipMemcpyAsync(cudaInsideVertices, insideVertices, sizeInsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken binnenste mesh naar GPU*/
	int3* insideTriangles = innerMesh->getInt3ArrayTriangles();
	int3* cudaInsideTriangles;
	int sizeInsideTriangles = numberOfInsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaInsideTriangles, sizeInsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaInsideTriangles, insideTriangles, sizeInsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren hoekpunten buitenste mesh naar GPU*/
	float3* outsideVertices = outerMesh->getFloat3ArrayVertices();
	float3* cudaOutsideVertices;
	int sizeOutsideVertices = numberOfOutsideVertices * sizeof(float3);
	handleCudaError(hipMalloc((void**)&cudaOutsideVertices, sizeOutsideVertices));
	handleCudaError(hipMemcpyAsync(cudaOutsideVertices, outsideVertices, sizeOutsideVertices, hipMemcpyHostToDevice));

	/* Alloceren en kopi�ren driehoeken buitenste mesh naar GPU*/
	int3* outsideTriangles = outerMesh->getInt3ArrayTriangles();
	int3* cudaOutsideTriangles;
	int sizeOutsideTriangles = numberOfOutsideTriangles * sizeof(int3);
	handleCudaError(hipMalloc((void**)&cudaOutsideTriangles, sizeOutsideTriangles));
	handleCudaError(hipMemcpyAsync(cudaOutsideTriangles, outsideTriangles, sizeOutsideTriangles, hipMemcpyHostToDevice));

	/* Alloceren voor mogelijke driehoeken die kunnen snijden uit BPCD op GPU*/
	int* intersectingTriangles;
	int sizeIntersectingTriangles = numberOfOutsideTriangles * 10 * sizeof(int);
	handleCudaError(hipMalloc((void**)&intersectingTriangles, sizeIntersectingTriangles));

	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now(); //stop time measurement
	auto transferDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	/* Als deze waarde > 0 ==> De binnenste mesh ligt niet volledig in de buitenste mesh*/
	int totalIntersections = 0;
	std::cout << "Kernel execution: TriangleTriangle_BlockPerOuterTriangle_BPCD" << std::endl;

	/*******************************************************************************
	Uitvoeren CUDA kernel - Triangle Triangle met Broad Phase Collision Detection
	********************************************************************************/
	/* Uitvoeren CUDA kernel*/
	hipEventRecord(start_event);
	triangle_triangle_GPU_BPCD_1_ThreadPerOuterTriangle << <(numberOfOutsideTriangles + 255) / 256, 256 >> > (cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, numberOfInsideTriangles, numberOfOutsideTriangles, intersectingTriangles);
	hipEventRecord(stop_event);

	hipError_t err = hipGetLastError();
	handleCudaError(err);
	hipEventSynchronize(stop_event);
	float milliseconds_1 = 0;
	hipEventElapsedTime(&milliseconds_1, start_event, stop_event);

	hipEventRecord(start_event);
	triangle_triangle_GPU_BPCD_2_BlockPerOuterTriangle << <numberOfOutsideTriangles, 128 >> > (cudaInsideTriangles, cudaInsideVertices, cudaOutsideTriangles, cudaOutsideVertices, cudaInside, intersectingTriangles);
	hipEventRecord(stop_event);

	err = hipGetLastError();
	handleCudaError(err);

	/* Kopi�ren van de resultaten van GPU naar CPU*/
	//handleCudaError(hipMemcpy(intersectionsPerInsideTriangle, cudaIntersectionsPerInsideTriangle, numberOfInsideTriangles * sizeof(int), hipMemcpyDeviceToHost));
	handleCudaError(hipMemcpy(inside, cudaInside, sizeof(bool), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop_event);
	float milliseconds_2 = 0;
	hipEventElapsedTime(&milliseconds_2, start_event, stop_event);

	std::string result;
	if (*inside) {
		result = "SNIJDEN NIET";
	}
	else {
		result = "SNIJDEN WEL";
	}
	std::cout << result << std::endl;
	output.push_back(std::to_string(milliseconds_1) + ";" + std::to_string(milliseconds_2) + ";" + std::to_string(milliseconds_1 + milliseconds_2) + ";" + result + ";" + std::to_string((float)transferDuration / 1000) + ";");


	hipFree(cudaInsideTriangles);
	hipFree(cudaInsideVertices);
	hipFree(cudaOutsideTriangles);
	hipFree(cudaOutsideVertices);
	hipFree(cudaInside);
	hipFree(intersectingTriangles);
	//hipFree(cudaIntersectionsPerInsideTriangle);
	hipHostFree(outsideTriangles);
	hipHostFree(outsideVertices);
	hipHostFree(insideTriangles);
	hipHostFree(insideVertices);
	//delete intersectionsPerInsideTriangle;

	delete inside;
}

void handleCudaError(hipError_t cudaERR) {
	if (cudaERR != hipSuccess) {
		printf("CUDA ERROR : %s\n", hipGetErrorString(cudaERR));
	}
}

__global__ void startGPU() {
	printf("GPU ready!\n");
}

void writeResultsToFile(std::vector<std::string>& result)
{
	std::vector<std::string>::iterator itr;
	std::string path = "output.csv";
	std::ofstream ofs;
	ofs.open(path, std::ofstream::out | std::ofstream::app);
	for (itr = result.begin(); itr != result.end(); ++itr)
	{
		ofs << (*itr);
	}
}
